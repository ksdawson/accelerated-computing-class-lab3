// Memory Coalescing Benchmark Kernels
//
// This file contains CUDA kernels to demonstrate and benchmark the performance
// difference between coalesced and non-coalesced memory access patterns:
// - Non-coalesced memory access pattern
// - Coalesced memory access pattern


#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>
#include <iostream>
#include <stdlib.h>
#include <vector>

using data_type = float;

#define THREADS_PER_WARP 32
#define WARPS 128
#define X 1024 // elements per thread

static constexpr size_t kNumOfOuterIterations = 5;
static constexpr size_t kNumOfInnerIterations = 3;

////////////////////////////////////////////////////////////////////////////////
// Non-Coalesced Memory Access Pattern

__global__ void non_coalesced_load(data_type *dst, data_type *src, int x) {
    // Each thread gets a contiguous block of x elements
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int start_block = thread_id * x;
    // Thread loads x elements strided by 1
    int start_index = start_block;
    int end_index = start_index + x;
    for (int i = start_index; i < end_index; ++i) {
        dst[i] = src[i];
    }
}

////////////////////////////////////////////////////////////////////////////////
// Coalesced Memory Access Pattern

__global__ void coalesced_load(data_type *dst, data_type *src, int x) {
    // Each block gets a contiguous block of x * blockDim.x elements
    int block_id = blockIdx.x;
    int start_block = block_id* (x * blockDim.x);
    // Thread loads x elements strided by blockDim.x
    int start_index = start_block + threadIdx.x;
    int end_index = start_index + x * blockDim.x;
    for (int i = start_index; i < end_index; i += blockDim.x) {
        dst[i] = src[i];
    }
}

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

// CUDA error checking macro
#define CUDA_CHECK(x) \
    do { \
        hipError_t err = (x); \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error " << static_cast<int>(err) << " (" \
                      << hipGetErrorString(err) << ") at " << __FILE__ << ":" \
                      << __LINE__ << std::endl; \
            std::exit(EXIT_FAILURE); \
        } \
    } while (0)

// BENCHPRESS macro for sophisticated benchmarking
#define BENCHPRESS(kernel_name, kNumOfOuterIterations, kNumOfInnerIterations, ...) \
    do { \
        std::cout << "Running " << #kernel_name << " ...\n"; \
        std::vector<double> times(kNumOfOuterIterations); \
        for (size_t i = 0; i < kNumOfOuterIterations; ++i) { \
            auto start = std::chrono::high_resolution_clock::now(); \
            for (size_t j = 0; j < kNumOfInnerIterations; ++j) { \
                kernel_name<<<WARPS, THREADS_PER_WARP>>>(__VA_ARGS__); \
            } \
            CUDA_CHECK(hipDeviceSynchronize()); \
            auto end = std::chrono::high_resolution_clock::now(); \
            times[i] = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start) \
                           .count() / \
                kNumOfInnerIterations; \
        } \
        std::sort(times.begin(), times.end()); \
        std::cout << "  Runtime: " << times[0] / 1'000'000 << " ms" << std::endl; \
    } while (0)

int main() {
    // Initialize data and allocate memory
    int total_threads = THREADS_PER_WARP * WARPS;
    int total_elements = total_threads * X;

    data_type *h_src = (data_type *)malloc(sizeof(data_type) * total_elements);
    data_type *h_dst = (data_type *)malloc(sizeof(data_type) * total_elements);

    for (int i = 0; i < total_elements; i++) {
        h_src[i] = static_cast<data_type>(i);
    }

    data_type *d_src = nullptr;
    data_type *d_dst = nullptr;
    CUDA_CHECK(hipMalloc(&d_src, total_elements * sizeof(data_type)));
    CUDA_CHECK(hipMalloc(&d_dst, total_elements * sizeof(data_type)));
    CUDA_CHECK(hipMemcpy(
        d_src,
        h_src,
        total_elements * sizeof(data_type),
        hipMemcpyHostToDevice));

    // Run benchmarks and test correctness
    CUDA_CHECK(hipMemset(d_dst, 0, total_elements * sizeof(data_type)));
    BENCHPRESS(
        non_coalesced_load,
        kNumOfOuterIterations,
        kNumOfInnerIterations,
        d_dst,
        d_src,
        X);

    CUDA_CHECK(hipMemcpy(
        h_dst,
        d_dst,
        total_elements * sizeof(data_type),
        hipMemcpyDeviceToHost));

    bool non_coalesced_correct = true;
    for (int i = 0; i < total_elements; i++) {
        if (h_dst[i] != h_src[i]) {
            non_coalesced_correct = false;
            break;
        }
    }
    std::cout << "non_coalesced_load: " << (non_coalesced_correct ? "PASSED" : "FAILED")
              << std::endl;

    CUDA_CHECK(hipMemset(d_dst, 0, total_elements * sizeof(data_type)));
    BENCHPRESS(
        coalesced_load,
        kNumOfOuterIterations,
        kNumOfInnerIterations,
        d_dst,
        d_src,
        X);

    CUDA_CHECK(hipMemcpy(
        h_dst,
        d_dst,
        total_elements * sizeof(data_type),
        hipMemcpyDeviceToHost));

    bool coalesced_correct = true;
    for (int i = 0; i < total_elements; i++) {
        if (h_dst[i] != h_src[i]) {
            coalesced_correct = false;
            break;
        }
    }
    std::cout << "coalesced_load: " << (coalesced_correct ? "PASSED" : "FAILED")
              << std::endl;

    // Clean up memory
    CUDA_CHECK(hipFree(d_src));
    CUDA_CHECK(hipFree(d_dst));
    free(h_src);
    free(h_dst);
}
