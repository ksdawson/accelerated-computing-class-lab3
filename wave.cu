#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <limits>
#include <utility>
#include <vector>

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Already Written)

// 'wave_cpu_step':
//
// Input:
//
//     t -- time coordinate
//     u(t - dt) in array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t) in array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     u(t + dt) in array 'u0' (overwrites the input)
//
template <typename Scene> void wave_cpu_step(float t, float *u0, float const *u1) {
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;
    constexpr float c = Scene::c;
    constexpr float dx = Scene::dx;
    constexpr float dt = Scene::dt;

    for (int32_t idx_y = 0; idx_y < n_cells_y; ++idx_y) {
        for (int32_t idx_x = 0; idx_x < n_cells_x; ++idx_x) {
            int32_t idx = idx_y * n_cells_x + idx_x;
            bool is_border =
                (idx_x == 0 || idx_x == n_cells_x - 1 || idx_y == 0 ||
                 idx_y == n_cells_y - 1);
            float u_next_val;
            if (is_border || Scene::is_wall(idx_x, idx_y)) {
                u_next_val = 0.0f;
            } else if (Scene::is_source(idx_x, idx_y)) {
                u_next_val = Scene::source_value(idx_x, idx_y, t);
            } else {
                constexpr float coeff = c * c * dt * dt / (dx * dx);
                float damping = Scene::damping(idx_x, idx_y);
                u_next_val =
                    ((2.0f - damping - 4.0f * coeff) * u1[idx] -
                     (1.0f - damping) * u0[idx] +
                     coeff *
                         (u1[idx - 1] + u1[idx + 1] + u1[idx - n_cells_x] +
                          u1[idx + n_cells_x]));
            }
            u0[idx] = u_next_val;
        }
    }
}

// 'wave_cpu':
//
// Input:
//
//     t0 -- initial time coordinate
//     n_steps -- number of time steps to simulate
//     u(t0 - dt) in array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t0) in array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     Overwrites contents of memory pointed to by 'u0' and 'u1'
//
//     Returns pointers to buffers containing the final states of the wave
//     u(t0 + (n_steps - 1) * dt) and u(t0 + n_steps * dt).
//
template <typename Scene>
std::pair<float *, float *> wave_cpu(float t0, int32_t n_steps, float *u0, float *u1) {
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step++) {
        float t = t0 + idx_step * Scene::dt;
        wave_cpu_step<Scene>(t, u0, u1);
        std::swap(u0, u1);
    }
    return {u0, u1};
}

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// GPU Implementation (Naive)

// 'wave_gpu_step':
//
// Input:
//
//     t -- time coordinate
//     u(t - dt) in GPU array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t) in GPU array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     u(t + dt) in GPU array 'u0' (overwrites the input)
//
template <typename Scene>
__global__ void wave_gpu_naive_step(
    float t,
    float *u0,      /* pointer to GPU memory */
    float const *u1, /* pointer to GPU memory */
    uint8_t ilp_size = 1
) {
    // Scene parameters
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;
    constexpr float c = Scene::c;
    constexpr float dx = Scene::dx;
    constexpr float dt = Scene::dt;

    // Thread info
    int tot_threads = gridDim.x * blockDim.x;
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Flatten the 2D iteration space into 1D and stride tot_threads pixels each iteration
    for (uint64_t idx = thread_index * ilp_size; idx < n_cells_y * n_cells_x; idx += tot_threads * ilp_size) {
        #pragma unroll
        for (uint8_t i = 0; i < ilp_size; ++i) {
            // Use 32x1 vectors
            uint64_t ilp_idx = idx + i;
            uint32_t idx_y = ilp_idx / n_cells_x;
            uint32_t idx_x = ilp_idx % n_cells_x;

            // Wave math
            bool is_border =
                (idx_x == 0 || idx_x == n_cells_x - 1 || idx_y == 0 ||
                    idx_y == n_cells_y - 1);
            float u_next_val;
            if (is_border || Scene::is_wall(idx_x, idx_y)) {
                u_next_val = 0.0f;
            } else if (Scene::is_source(idx_x, idx_y)) {
                u_next_val = Scene::source_value(idx_x, idx_y, t);
            } else {
                constexpr float coeff = c * c * dt * dt / (dx * dx);
                float damping = Scene::damping(idx_x, idx_y);
                u_next_val =
                    ((2.0f - damping - 4.0f * coeff) * u1[ilp_idx] -
                        (1.0f - damping) * u0[ilp_idx] +
                        coeff *
                            (u1[ilp_idx - 1] + u1[ilp_idx + 1] + u1[ilp_idx - n_cells_x] +
                            u1[ilp_idx + n_cells_x]));
            }
            u0[ilp_idx] = u_next_val;
        }
    }
}

// 'wave_gpu_naive':
//
// Input:
//
//     t0 -- initial time coordinate
//     n_steps -- number of time steps to simulate
//     u(t0 - dt) in GPU array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t0) in GPU array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     Launches kernels to overwrite the GPU memory pointed to by 'u0' and 'u1'
//
//     Returns pointers to GPU buffers which will contain the final states of
//     the wave u(t0 + (n_steps - 1) * dt) and u(t0 + n_steps * dt) after all
//     launched kernels have completed.
//
template <typename Scene>
std::pair<float *, float *> wave_gpu_naive(
    float t0,
    int32_t n_steps,
    float *u0, /* pointer to GPU memory */
    float *u1  /* pointer to GPU memory */
) {
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step++) {
        float t = t0 + idx_step * Scene::dt;
        wave_gpu_naive_step<Scene><<<48, 32 * 32>>>(t, u0, u1);
        std::swap(u0, u1);
    }
    return {u0, u1};
}

////////////////////////////////////////////////////////////////////////////////
// GPU Implementation (With Shared Memory)

// Helper to handle shrinking the valid data tile
// __device__ void shrink_valid_data_tile(Tile *valid_data_tile) {
//     // Get scene and sm tiles
//     Tile *sm_tile = valid_data_tile->parent;
//     Tile *scene_tile = sm_tile->parent;

//     // Get location of the valid data tile in the scene
//     uint32_t first_sm_idx = transform_to_parent_idx(valid_data_tile, 0);
//     uint32_t last_sm_idx = transform_to_parent_idx(valid_data_tile, valid_data_tile->height * valid_data_tile->width - 1);
//     uint32_t first_scene_idx = transform_to_parent_idx(sm_tile, first_sm_idx);
//     uint32_t last_scene_idx = transform_to_parent_idx(sm_tile, last_sm_idx);

//     // Get the bounds of the tile in the scene
//     uint32_t first_scene_idx_y = scene_tile->get_idx_y(first_scene_idx);
//     uint32_t first_scene_idx_x = scene_tile->get_idx_x(first_scene_idx);
//     uint32_t last_scene_idx_y = scene_tile->get_idx_y(last_scene_idx);
//     uint32_t last_scene_idx_x = scene_tile->get_idx_x(last_scene_idx);

//     // Move the valid data tile in the sm tile
//     if (first_scene_idx_y != 0) {
//         // Move over a col in the parent
//         valid_data_tile->idx_in_parent += sm_tile->height;
//     }
//     if (first_scene_idx_x != 0) {
//         // Move down a row in the parent
//         valid_data_tile->idx_in_parent += 1;
//     }

//     // Determine if at an edge
//     bool column_edge = first_scene_idx_y == 0 || last_scene_idx_y == scene_tile->width - 1;
//     bool row_edge = first_scene_idx_x == 0 || last_scene_idx_x == scene_tile->height - 1;
//     // Shrink the tile
//     valid_data_tile->height -= column_edge ? 1 : 2;
//     valid_data_tile->width -= row_edge ? 1 : 2;
// }

// template <typename Scene>
// __global__ void wave_gpu_shmem_multistep(
//     float t0, uint32_t ti_step, uint32_t tf_step, // Time params
//     float *u0, float *u1 // Buffer params
// ) {
//     // Scene dimensions
//     uint32_t scene_height = Scene::n_cells_x;
//     uint32_t scene_width = Scene::n_cells_y;

//     // Valid data location in the scene and SM tile
//     uint32_t scene_idx;
//     uint32_t sm_idx = 0;

//     // SM tile dimensions
//     uint32_t sm_height, sm_width;
//     setup_sm_tile(scene_height, scene_width, tf_step - ti_step - 1, &sm_height, &sm_width, &scene_idx);

//     //
//     uint32_t scene_idx_y = scene_idx / scene_height;
//     uint32_t scene_idx_x = scene_idx % scene_height;
//     uint32_t sm_idx_y = 0;
//     uint32_t sm_idx_x = 0;

//     // Valid data dimensions
//     uint32_t valid_data_height = sm_height;
//     uint32_t valid_data_width = sm_width;

//     // Setup the block's SRAM
//     extern __shared__ float sram[];
//     // Create SM tile size arrays in SRAM
//     float *u0_local = sram;
//     float *u1_local = sram + sm_height * sm_width;
//     // Load data from main memory
//     load_shmem(u0, u1, u0_local, u1_local, scene_height, scene_idx, sm_height, sm_width);
    
//     // Debug
//     // if (threadIdx.x == 0) {
//     //     printf("b, h, w, sidx, smidx: %d, %u, %u, %u, %u\n", blockIdx.x, sm_height, sm_width, scene_idx, sm_idx);
//     // }

//     // Iterate over the time steps
//     for (uint32_t idx_step = ti_step; idx_step < tf_step; ++idx_step) {
//         // Calculate t
//         float t = t0 + idx_step * Scene::dt;

//         // Flatten the 2D iteration space into 1D and stride tot_threads pixels each iteration
//         for (uint64_t valid_data_idx = threadIdx.x; valid_data_idx < valid_data_height * valid_data_width; valid_data_idx += blockDim.x) {
//             // Valid data location
//             uint32_t valid_data_idx_y = valid_data_idx / valid_data_height;
//             uint32_t valid_data_idx_x = valid_data_idx % valid_data_height;
//             // Map to SM location
//             uint32_t curr_sm_idx = (sm_idx_y + valid_data_idx_y) * sm_height + valid_data_idx_x;
//             // Map to scene location
//             uint32_t curr_scene_idx = (scene_idx_y + valid_data_idx_y) * scene_height + valid_data_idx_x;
//             // Wave math: Calculations use scene idx and memory uses sm idx
//             wave<Scene>(curr_scene_idx, t, u0_local, u1_local, curr_sm_idx, sm_height);
//         }

//         // We need the new pixel for all pixels in the block before processing the next time step
//         __syncthreads();

//         if (idx_step < tf_step - 1) {
//             // u0 contains the most recent timestamp and u1 contains the second most recent so swap
//             std::swap(u0_local, u1_local); // Only swaps pointers in local registers
//             // TODO: Shrink the valid data tile
//         }
//     }

//     // Store data to main memory
//     std::swap(u0_local, u1_local);
//     store_shmem(u0, u1, u0_local, u1_local, scene_height, scene_idx, sm_height, sm_idx, valid_data_height, valid_data_width);
// }

// Helpers to load/store data
__device__ void load_shmem(
    float *u0, float *u1, // Main memory buffer params
    float *u0_local, float *u1_local, // SRAM buffer params
    uint32_t scene_height, uint32_t scene_idx_y, uint32_t scene_idx_x, // Scene params
    uint32_t sm_height, uint32_t sm_width // SM tile params
) {
    // Load data from main memory
    for (uint64_t sm_idx = threadIdx.x; sm_idx < sm_height * sm_width; sm_idx += blockDim.x) {
        // Map SM to scene idx
        uint32_t scene_idx = (scene_idx_y + sm_idx / sm_height) * scene_height + (scene_idx_x + sm_idx % sm_height);
        // Copy memory over
        u0_local[sm_idx] = u0[scene_idx];
        u1_local[sm_idx] = u1[scene_idx];
    }
    // Wait for all the memory to be loaded
    __syncthreads();
}
__device__ void store_shmem(
    float *u0, float *u1, // Main memory buffer params
    float *u0_local, float *u1_local, // SRAM buffer params
    uint32_t scene_height, uint32_t scene_idx_y, uint32_t scene_idx_x, // Scene params
    uint32_t sm_height, uint32_t sm_idx_y, uint32_t sm_idx_x, // SM tile params
    uint32_t valid_data_height, uint32_t valid_data_width // Valid data tile params
) {
    // Store data to main memory
    for (uint64_t valid_data_idx = threadIdx.x; valid_data_idx < valid_data_height * valid_data_width; valid_data_idx += blockDim.x) {
        // Map valid data to SM idx
        uint32_t sm_idx = (sm_idx_y + valid_data_idx / valid_data_height) * sm_height + (sm_idx_x + valid_data_idx % valid_data_height);
        // Map valid data to scene idx
        uint32_t scene_idx = (scene_idx_y + valid_data_idx / valid_data_height) * scene_height + (scene_idx_x + valid_data_idx % valid_data_height);
        // Copy memory over
        u0[scene_idx] = u0_local[sm_idx];
        u1[scene_idx] = u1_local[sm_idx];
    }
    // Don't need to wait for all the memory to be stored since the whole kernel is synchronized?
    __syncthreads();
}

// Helper to setup the SM tile
__device__ void setup_sm_tile(uint32_t scene_height, uint32_t scene_width, uint8_t tile_expansion, // Input
    uint32_t *out_sm_height, uint32_t *out_sm_width, uint32_t *out_scene_idx_y, uint32_t *out_scene_idx_x, // Output
    uint8_t *height_shrink_amt, uint8_t *width_shrink_amt, uint8_t *idx_y_shrink_amt, uint8_t *idx_x_shrink_amt // Output
) {
    // Tile dimensions
    uint8_t tiles_per_col = 8; // Tuning parameter: We want as square as possible tiles?
    uint8_t tiles_per_row = gridDim.x / tiles_per_col;

    // Tile coordinates
    uint8_t tile_j = blockIdx.x / tiles_per_col;
    uint8_t tile_i = blockIdx.x % tiles_per_col;

    // Divide the scene into tiles (valid data that must be written back at the end)
    uint32_t tile_height = scene_height / tiles_per_col;
    uint32_t tile_width = scene_width / tiles_per_row;

    // Calculate starting scene idx of the tile
    uint32_t scene_idx_y = tile_j * tile_width;
    uint32_t scene_idx_x = tile_i * tile_height;

    // Handle grids not divisible by the number of SMs
    uint8_t extra_rows = scene_height % tiles_per_col;
    uint8_t extra_cols = scene_width % tiles_per_row;

    // Assign the extra to the edges since they have smaller overlap (limit to last for simplicity)
    tile_width += (tile_j == tiles_per_row - 1) ? extra_cols : 0;
    tile_height += (tile_i == tiles_per_col - 1) ? extra_rows : 0;

    // Expand the tile by the number of time steps in each direction (overlap for invalid data)
    // Note, edges can only expand in one dir
    tile_width += (tile_j == 0 || tile_j == tiles_per_row - 1) ? tile_expansion : 2 * tile_expansion;
    tile_height += (tile_i == 0 || tile_i == tiles_per_col - 1) ? tile_expansion : 2 * tile_expansion;

    // Set the SM tile
    *out_sm_height = tile_height;
    *out_sm_width = tile_width;
    *out_scene_idx_y = scene_idx_y;
    *out_scene_idx_x = scene_idx_x;

    // Set the shrink amts
    *width_shrink_amt = (tile_j == 0 || tile_j == tiles_per_row - 1) ? 1 : 2;
    *height_shrink_amt = (tile_i == 0 || tile_i == tiles_per_col - 1) ? 1 : 2;
    *idx_y_shrink_amt = (tile_j == 0) ? 0 : 1;
    *idx_x_shrink_amt = (tile_i == 0) ? 0 : 1;
}

template <typename Scene>
__device__ void wave(uint32_t idx_y, uint32_t idx_x, float t, // Scene params
    float *u0, float *u1, // Buffer params
    uint32_t memory_idx, uint32_t memory_height // Memory params
) {
    // Scene parameters
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;
    constexpr float c = Scene::c;
    constexpr float dx = Scene::dx;
    constexpr float dt = Scene::dt;

    // Wave math
    bool is_border =
        (idx_x == 0 || idx_x == n_cells_x - 1 || idx_y == 0 ||
            idx_y == n_cells_y - 1);
    float u_next_val;
    if (is_border || Scene::is_wall(idx_x, idx_y)) {
        u_next_val = 0.0f;
    } else if (Scene::is_source(idx_x, idx_y)) {
        u_next_val = Scene::source_value(idx_x, idx_y, t);
    } else {
        constexpr float coeff = c * c * dt * dt / (dx * dx);
        float damping = Scene::damping(idx_x, idx_y);
        u_next_val =
            ((2.0f - damping - 4.0f * coeff) * u1[memory_idx] -
                (1.0f - damping) * u0[memory_idx] +
                coeff *
                    (u1[memory_idx - 1] + u1[memory_idx + 1] + u1[memory_idx - memory_height] +
                    u1[memory_idx + memory_height]));
    }
    u0[memory_idx] = u_next_val;
}

template <typename Scene>
__global__ void wave_gpu_shmem_multistep(
    float t0, uint32_t ti_step, uint32_t tf_step, // Time params
    float *u0, float *u1 // Buffer params
) {
    // Scene dimensions
    uint32_t scene_height = Scene::n_cells_x;
    uint32_t scene_width = Scene::n_cells_y;
    // Valid data location in the scene and SM tile
    uint32_t scene_idx_y, scene_idx_x;
    uint32_t sm_idx_y = 0;
    uint32_t sm_idx_x = 0;
    // Shrink parameters
    uint8_t height_shrink_amt, width_shrink_amt, idx_y_shrink_amt, idx_x_shrink_amt;
    // SM tile dimensions
    uint32_t sm_height, sm_width;
    // setup_sm_tile(scene_height, scene_width, tf_step - ti_step - 1, &sm_height, &sm_width, &scene_idx_y, &scene_idx_x,
    //     &height_shrink_amt, &width_shrink_amt, &idx_y_shrink_amt, &idx_x_shrink_amt
    // );
    setup_sm_tile(scene_height, scene_width, tf_step - ti_step, &sm_height, &sm_width, &scene_idx_y, &scene_idx_x,
        &height_shrink_amt, &width_shrink_amt, &idx_y_shrink_amt, &idx_x_shrink_amt
    );
    // Valid data dimensions
    uint32_t valid_data_height = sm_height;
    uint32_t valid_data_width = sm_width;

    // Setup the block's SRAM
    extern __shared__ float sram[];
    // Create SM tile size arrays in SRAM
    float *u0_local = sram;
    float *u1_local = sram + sm_height * sm_width;
    // Load data from main memory
    load_shmem(u0, u1, u0_local, u1_local,
        scene_height, scene_idx_y, scene_idx_x,
        sm_height, sm_width
    );

    // Verify load was correct -> CORRECT
    for (uint64_t valid_data_idx = threadIdx.x; valid_data_idx < valid_data_height * valid_data_width; valid_data_idx += blockDim.x) {
        uint32_t sm_idx = (sm_idx_y + valid_data_idx / valid_data_height) * sm_height + (sm_idx_x + valid_data_idx % valid_data_height);
        uint32_t scene_idx = (scene_idx_y + valid_data_idx / valid_data_height) * scene_height + (scene_idx_x + valid_data_idx % valid_data_height);
        if (u0[scene_idx] != u0_local[sm_idx]) {
            printf("(u0 loadn step %u) global, local: %f, %f\n", ti_step, u0[scene_idx], u0_local[sm_idx]);
            return;
        }
        if (u1[scene_idx] != u1_local[sm_idx]) {
            printf("(u1 load step %u) global, local: %f, %f\n", ti_step, u1[scene_idx], u1_local[sm_idx]);
            return;
        }
    }

    // Iterate over the time steps
    for (uint32_t idx_step = ti_step; idx_step < tf_step; ++idx_step) {
        // Calculate t
        float t = t0 + idx_step * Scene::dt;

        // Shrink the tile
        valid_data_height -= height_shrink_amt;
        valid_data_width -= width_shrink_amt;
        // scene_idx_y += idx_y_shrink_amt;
        // scene_idx_x += idx_x_shrink_amt;
        sm_idx_y += idx_y_shrink_amt;
        sm_idx_x += idx_x_shrink_amt;

        // Flatten the 2D iteration space into 1D and stride tot_threads pixels each iteration
        for (uint64_t valid_data_idx = threadIdx.x; valid_data_idx < valid_data_height * valid_data_width; valid_data_idx += blockDim.x) {
            // Map valid data to SM idx
            uint32_t sm_idx = (sm_idx_y + valid_data_idx / valid_data_height) * sm_height + (sm_idx_x + valid_data_idx % valid_data_height);
            // Map valid data idx to scene idx
            uint32_t new_scene_idx_y = scene_idx_y + valid_data_idx / valid_data_height;
            uint32_t new_scene_idx_x = scene_idx_x + valid_data_idx % valid_data_height;
            // uint32_t scene_idx = new_scene_idx_y * scene_height + new_scene_idx_x;

            // Wave math: Calculations use scene idx and memory uses sm idx
            // wave<Scene>(new_scene_idx_y, new_scene_idx_x, t,
            //     u0, u1,
            //     scene_idx, scene_height
            // );
            wave<Scene>(new_scene_idx_y, new_scene_idx_x, t,
                u0_local, u1_local,
                sm_idx, sm_height
            );

            // Verify wave was correct -> INCORRECT
            // I think the problem is that a tile has no neighbors on the edges!
            // if (u0[scene_idx] != u0_local[sm_idx]) {
            //     printf("(u0 wave2 step %u) global, local: %f, %f\n", idx_step, u0[scene_idx], u0_local[sm_idx]);
            //     return;
            // }
            // if (u1[scene_idx] != u1_local[sm_idx]) {
            //     printf("(u1 wave2 step %u) global, local: %f, %f\n", idx_step, u1[scene_idx], u1_local[sm_idx]);
            //     return;
            // }
        }

        // We need the new pixel for all pixels in the block before processing the next time step
        __syncthreads();

        // u0 contains the most recent timestamp and u1 contains the second most recent so swap
        // std::swap(u0, u1); // Only swaps pointers in local registers
        // std::swap(u0_local, u1_local);
    }

    // Store data to main memory
    store_shmem(u0, u1, u0_local, u1_local,
        scene_height, scene_idx_y, scene_idx_x,
        sm_height, sm_idx_y, sm_idx_x,
        valid_data_height, valid_data_width
    );

    // Verify store was correct -> CORRECT
    for (uint64_t valid_data_idx = threadIdx.x; valid_data_idx < valid_data_height * valid_data_width; valid_data_idx += blockDim.x) {
        uint32_t sm_idx = (sm_idx_y + valid_data_idx / valid_data_height) * sm_height + (sm_idx_x + valid_data_idx % valid_data_height);
        uint32_t scene_idx = (scene_idx_y + valid_data_idx / valid_data_height) * scene_height + (scene_idx_x + valid_data_idx % valid_data_height);
        if (u0[scene_idx] != u0_local[sm_idx]) {
            printf("(u0 store step %u) global, local: %f, %f\n", ti_step, u0[scene_idx], u0_local[sm_idx]);
            return;
        }
        if (u1[scene_idx] != u1_local[sm_idx]) {
            printf("(u1 store step %u) global, local: %f, %f\n", ti_step, u1[scene_idx], u1_local[sm_idx]);
            return;
        }
    }
}

// 'wave_gpu_shmem':
//
// Input:
//
//     t0 -- initial time coordinate
//
//     n_steps -- number of time steps to simulate
//
//     u(t0 - dt) in GPU array 'u0' of size 'n_cells_y * n_cells_x'
///
//     u(t0) in GPU array 'u1' of size 'n_cells_y * n_cells_x'
//
//     Scratch buffers 'extra0' and 'extra1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     Launches kernels to (potentially) overwrite the GPU memory pointed to
//     by 'u0' and 'u1', 'extra0', and 'extra1'.
//
//     Returns pointers to GPU buffers which will contain the final states of
//     the wave u(t0 + (n_steps - 1) * dt) and u(t0 + n_steps * dt) after all
//     launched kernels have completed. These buffers can be any of 'u0', 'u1',
//     'extra0', or 'extra1'.
//
template <typename Scene>
std::pair<float *, float *> wave_gpu_shmem(
    float t0,
    int32_t n_steps,
    float *u0,     /* pointer to GPU memory */
    float *u1,     /* pointer to GPU memory */
    float *extra0, /* pointer to GPU memory */
    float *extra1  /* pointer to GPU memory */
) {
    // Number of time steps to process at once in a kernel
    uint8_t time_steps = 1;

    for (uint32_t idx_step = 0; idx_step < n_steps; idx_step += time_steps) {
        // Compute starting and ending time step
        uint32_t ti_step = idx_step;
        uint32_t tf_step = ti_step + min(n_steps - idx_step, time_steps);

        // Setup the block SRAM
        int shmem_size_bytes = 100 * 1000; // Max 100 KB per block
        CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
            wave_gpu_shmem_multistep<Scene>),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            shmem_size_bytes
        ));

        // Launch our kernel
        wave_gpu_shmem_multistep<Scene><<<48, 32 * 32, shmem_size_bytes>>>(t0, ti_step, tf_step, u0, u1);

        // if (idx_step == 10) {
        //     return {u0, u1};
        // }

        // Treat the multi step kernel as one step so u0 will now contain the most recent
        std::swap(u0, u1);
    }
    return {u0, u1};
}

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

struct BaseScene {
    constexpr static int32_t n_cells_x = 3201;
    constexpr static int32_t n_cells_y = 3201;
    constexpr static float c = 1.0f;
    constexpr static float dx = 1.0f / float(n_cells_x - 1);
    constexpr static float dy = 1.0f / float(n_cells_y - 1);
    constexpr static float dt = 0.25f * dx / c;
    constexpr static float t_end = 1.0f;
};

struct BaseSceneSmallScale {
    constexpr static int32_t n_cells_x = 201;
    constexpr static int32_t n_cells_y = 201;
    constexpr static float c = 1.0f;
    constexpr static float dx = 1.0f / float(n_cells_x - 1);
    constexpr static float dy = 1.0f / float(n_cells_y - 1);
    constexpr static float dt = 0.25f * dx / c;
    constexpr static float t_end = 1.0f;
};

float __host__ __device__ __forceinline__ boundary_damping(
    int32_t n_cells_x,
    int32_t n_cells_y,
    float ramp_size,
    float max_damping,
    int32_t idx_x,
    int32_t idx_y) {
    float x = float(idx_x) / (n_cells_x - 1);
    float y = float(idx_y) / (n_cells_y - 1);
    float fx = 1.0f - min(min(x, 1.0f - x), ramp_size) / ramp_size;
    float fy = 1.0f - min(min(y, 1.0f - y), ramp_size) / ramp_size;
    float f = max(fx, fy);
    return max_damping * f * f;
}

struct PointSource : public BaseScene {
    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        return false;
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 2 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 20.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

struct Slit : public BaseScene {
    constexpr static float slit_width = 0.05f;

    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        float y = float(idx_y) / (n_cells_y - 1);
        return idx_x == (n_cells_x - 1) / 2 &&
            (y < 0.5f - slit_width / 2 || y > 0.5f + slit_width / 2);
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 4 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 40.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

struct DoubleSlit : public BaseScene {
    constexpr static float slit_width = 0.03f;

    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        float y = float(idx_y) / (n_cells_y - 1);
        return (idx_x == (n_cells_x - 1) * 2 / 3) &&
            !((y >= 0.45f - slit_width / 2 && y <= 0.45f + slit_width / 2) ||
              (y >= 0.55f - slit_width / 2 && y <= 0.55f + slit_width / 2));
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 6 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 20.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

struct DoubleSlitSmallScale : public BaseSceneSmallScale {
    constexpr static float slit_width = 0.03f;

    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        constexpr float EPS = 1e-6;
        float y = float(idx_y) / (n_cells_y - 1);
        return (idx_x == (n_cells_x - 1) * 2 / 3) &&
            !((y >= 0.45f - slit_width / 2 - EPS && y <= 0.45f + slit_width / 2 + EPS) ||
              (y >= 0.55f - slit_width / 2 - EPS && y <= 0.55f + slit_width / 2 + EPS));
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 6 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 20.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

// Output image writers: BMP file header structure
#pragma pack(push, 1)
struct BMPHeader {
    uint16_t fileType{0x4D42};   // File type, always "BM"
    uint32_t fileSize{0};        // Size of the file in bytes
    uint16_t reserved1{0};       // Always 0
    uint16_t reserved2{0};       // Always 0
    uint32_t dataOffset{54};     // Start position of pixel data
    uint32_t headerSize{40};     // Size of this header (40 bytes)
    int32_t width{0};            // Image width in pixels
    int32_t height{0};           // Image height in pixels
    uint16_t planes{1};          // Number of color planes
    uint16_t bitsPerPixel{24};   // Bits per pixel (24 for RGB)
    uint32_t compression{0};     // Compression method (0 for uncompressed)
    uint32_t imageSize{0};       // Size of raw bitmap data
    int32_t xPixelsPerMeter{0};  // Horizontal resolution
    int32_t yPixelsPerMeter{0};  // Vertical resolution
    uint32_t colorsUsed{0};      // Number of colors in the color palette
    uint32_t importantColors{0}; // Number of important colors
};
#pragma pack(pop)

void writeBMP(
    const char *fname,
    uint32_t width,
    uint32_t height,
    const std::vector<uint8_t> &pixels) {
    BMPHeader header;
    header.width = width;
    header.height = height;

    uint32_t rowSize = (width * 3 + 3) & (~3); // Align to 4 bytes
    header.imageSize = rowSize * height;
    header.fileSize = header.dataOffset + header.imageSize;

    std::ofstream file(fname, std::ios::binary);
    file.write(reinterpret_cast<const char *>(&header), sizeof(header));

    // Write pixel data with padding
    std::vector<uint8_t> padding(rowSize - width * 3, 0);
    for (int32_t idx_y = height - 1; idx_y >= 0;
         --idx_y) { // BMP stores pixels from bottom to top
        const uint8_t *row = &pixels[idx_y * width * 3];
        file.write(reinterpret_cast<const char *>(row), width * 3);
        if (!padding.empty()) {
            file.write(reinterpret_cast<const char *>(padding.data()), padding.size());
        }
    }
}

// If trunc - cut the border of the image.
template <typename Scene>
std::vector<uint8_t> render_wave(const float *u, int trunc = 0) {
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;

    std::vector<uint8_t> pixels((n_cells_x - trunc) * (n_cells_y - trunc) * 3);
    for (int32_t idx_y = 0; idx_y < n_cells_y - trunc; ++idx_y) {
        for (int32_t idx_x = 0; idx_x < n_cells_x - trunc; ++idx_x) {
            int32_t idx = idx_y * (n_cells_x - trunc) + idx_x;
            int32_t u_idx = idx_y * n_cells_x + idx_x;
            float val = u[u_idx];
            bool is_wall = Scene::is_wall(idx_x, idx_y);
            // BMP stores pixels in BGR order
            if (is_wall) {
                pixels[idx * 3 + 2] = 0;
                pixels[idx * 3 + 1] = 0;
                pixels[idx * 3 + 0] = 0;
            } else if (val > 0.0f) {
                pixels[idx * 3 + 2] = 255;
                pixels[idx * 3 + 1] = 255 - uint8_t(min(val * 255.0f, 255.0f));
                pixels[idx * 3 + 0] = 255 - uint8_t(min(val * 255.0f, 255.0f));
            } else {
                pixels[idx * 3 + 2] = 255 - uint8_t(min(-val * 255.0f, 255.0f));
                pixels[idx * 3 + 1] = 255 - uint8_t(min(-val * 255.0f, 255.0f));
                pixels[idx * 3 + 0] = 255;
            }
        }
    }
    return pixels;
}

struct Results {
    std::vector<float> u0_final;
    std::vector<float> u1_final;
    double time_ms;
};

template <typename Scene, typename F>
Results run_cpu(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    F func) {
    auto u0 = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u1 = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);

    std::pair<float *, float *> u_final;

    double best_time = std::numeric_limits<double>::infinity();
    for (int32_t i = 0; i < num_iters_outer; ++i) {
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t j = 0; j < num_iters_inner; ++j) {
            std::fill(u0.begin(), u0.end(), 0.0f);
            std::fill(u1.begin(), u1.end(), 0.0f);
            u_final = func(t0, n_steps, u0.data(), u1.data());
        }
        auto end = std::chrono::high_resolution_clock::now();
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count() /
            num_iters_inner;
        best_time = std::min(best_time, time_ms);
    }

    if (u_final.first == u1.data() && u_final.second == u0.data()) {
        std::swap(u0, u1);
    } else if (!(u_final.first == u0.data() && u_final.second == u1.data())) {
        std::cerr << "Unexpected return values from 'func'" << std::endl;
        std::abort();
    }

    return {std::move(u0), std::move(u1), best_time};
}

template <typename Scene, typename F>
Results run_gpu(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    bool use_extra,
    F func) {
    float *u0;
    float *u1;
    float *extra0 = nullptr;
    float *extra1 = nullptr;

    CUDA_CHECK(hipMalloc(&u0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&u1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));

    if (use_extra) {
        CUDA_CHECK(
            hipMalloc(&extra0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
        CUDA_CHECK(
            hipMalloc(&extra1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    }

    std::pair<float *, float *> u_final;

    double best_time = std::numeric_limits<double>::infinity();
    for (int32_t i = 0; i < num_iters_outer; ++i) {
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t j = 0; j < num_iters_inner; ++j) {
            CUDA_CHECK(
                hipMemset(u0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
            CUDA_CHECK(
                hipMemset(u1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
            if (use_extra) {
                CUDA_CHECK(hipMemset(
                    extra0,
                    0,
                    Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
                CUDA_CHECK(hipMemset(
                    extra1,
                    0,
                    Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
            }
            u_final = func(t0, n_steps, u0, u1, extra0, extra1);
        }
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count() /
            num_iters_inner;
        best_time = std::min(best_time, time_ms);
    }

    if (u_final.first != u0 && u_final.first != u1 &&
        (extra0 == nullptr || u_final.first != extra0) &&
        (extra1 == nullptr || u_final.first != extra1)) {
        std::cerr << "Unexpected final 'u0' pointer returned from GPU implementation"
                  << std::endl;
        std::abort();
    }

    if (u_final.second != u0 && u_final.second != u1 &&
        (extra0 == nullptr || u_final.second != extra0) &&
        (extra1 == nullptr || u_final.second != extra1)) {
        std::cerr << "Unexpected final 'u1' pointer returned from GPU implementation"
                  << std::endl;
        std::abort();
    }

    auto u0_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u1_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    CUDA_CHECK(hipMemcpy(
        u0_cpu.data(),
        u_final.first,
        Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
        hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(
        u1_cpu.data(),
        u_final.second,
        Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
        hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(u0));
    CUDA_CHECK(hipFree(u1));
    if (use_extra) {
        CUDA_CHECK(hipFree(extra0));
        CUDA_CHECK(hipFree(extra1));
    }

    return {std::move(u0_cpu), std::move(u1_cpu), best_time};
}

template <typename Scene, typename F>
Results run_gpu_extra(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    F func) {
    return run_gpu<Scene>(t0, n_steps, num_iters_outer, num_iters_inner, true, func);
}

template <typename Scene, typename F>
Results run_gpu_no_extra(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    F func) {
    return run_gpu<Scene>(
        t0,
        n_steps,
        num_iters_outer,
        num_iters_inner,
        false,
        [&](float t0,
            int32_t n_steps,
            float *u0,
            float *u1,
            float *extra0,
            float *extra1) { return func(t0, n_steps, u0, u1); });
}

double rel_rmse(std::vector<float> const &a, std::vector<float> const &b) {
    if (a.size() != b.size()) {
        std::cerr << "Mismatched sizes in 'rel_rmse'" << std::endl;
        std::abort();
    }
    double ref_sum = 0.0;
    double sum = 0.0;
    for (size_t i = 0; i < a.size(); ++i) {
        ref_sum += double(a.at(i)) * double(a.at(i));
        double diff = double(a.at(i)) - double(b.at(i));
        sum += diff * diff;
    }
    return sqrt(sum / a.size()) / sqrt(ref_sum / a.size());
}

// FFmpeg implementations.
typedef std::vector<std::vector<uint8_t>> FFmpegFrames;

// CPU implementation with FFmpeg framing.
template <typename Scene>
void wave_ffmpeg(float t0, int32_t n_steps, FFmpegFrames &frames) {
    static constexpr int32_t frame_step = 2;
    auto u0_v = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u1_v = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u0 = u0_v.data();
    auto u1 = u1_v.data();
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step += frame_step) {
        auto r = wave_cpu<Scene>(t0 + idx_step * Scene::dt, frame_step, u0, u1);
        u0 = r.first;
        u1 = r.second;
        frames.push_back(render_wave<Scene>(u1, 1));
    }
}

template <typename Scene>
void wave_ffmpeg_gpu(float t0, int32_t n_steps, FFmpegFrames &frames) {
    static constexpr int32_t frame_step = 2;
    auto u1_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    float *u0;
    float *u1;
    CUDA_CHECK(hipMalloc(&u0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&u1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step += frame_step) {
        auto r = wave_gpu_naive<Scene>(t0 + idx_step * Scene::dt, frame_step, u0, u1);
        u0 = r.first;
        u1 = r.second;
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(
            u1_cpu.data(),
            u1,
            Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
            hipMemcpyDeviceToHost));
        frames.push_back(render_wave<Scene>(u1_cpu.data(), 1));
    }
}

template <typename Scene>
void wave_ffmpeg_gpu_shmem(float t0, int32_t n_steps, FFmpegFrames &frames) {
    static constexpr int32_t frame_step = 2;
    auto u1_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    float *u0;
    float *u1;
    float *extra0;
    float *extra1;
    CUDA_CHECK(hipMalloc(&u0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&u1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&extra0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&extra1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    float *buffers[] = {u0, u1, extra0, extra1};
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step += frame_step) {
        CUDA_CHECK(
            hipMemset(extra0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
        CUDA_CHECK(
            hipMemset(extra1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
        auto r = wave_gpu_shmem<Scene>(
            t0 + idx_step * Scene::dt,
            frame_step,
            u0,
            u1,
            extra0,
            extra1);
        u0 = r.first;
        u1 = r.second;
        for (int i = 0; i < 4; ++i) {
            if (buffers[i] != u0 && buffers[i] != u1) {
                extra0 = buffers[i];
            }
        }
        for (int i = 0; i < 4; ++i) {
            if (buffers[i] != u0 && buffers[i] != u1 && buffers[i] != extra0) {
                extra1 = buffers[i];
            }
        }
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(
            u1_cpu.data(),
            u1,
            Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
            hipMemcpyDeviceToHost));
        frames.push_back(render_wave<Scene>(u1_cpu.data(), 1));
    }
}

template <typename Scene>
int generate_animation(const FFmpegFrames &frames, std::string fname) {
    std::string ffmpeg_command = "ffmpeg -y "
                                 "-f rawvideo "
                                 "-pixel_format rgb24 "
                                 "-video_size " +
        std::to_string(Scene::n_cells_x - 1) + "x" +
        std::to_string(Scene::n_cells_y - 1) +
        " "
        "-framerate " +
        std::to_string(30) +
        " "
        "-i - "
        "-c:v libx264 "
        "-pix_fmt yuv420p " +
        fname + ".mp4" + " 2> /dev/null";

    FILE *pipe = popen(ffmpeg_command.c_str(), "w");
    if (!pipe) {
        std::cerr << "Failed to open pipe to FFmpeg." << std::endl;
        return 1;
    }

    for (auto &frame : frames) {
        if (fwrite(frame.data(), 1, frame.size(), pipe) != frame.size()) {
            std::cerr << "Failed to write frame to FFmpeg." << std::endl;
            return 1;
        }
    }

    pclose(pipe);
    return 0;
}

int main(int argc, char **argv) {
    // Small scale tests: mainly for correctness.
    double tolerance = 3e-2;
    bool gpu_naive_correct = false;
    bool gpu_shmem_correct = false;
    {
        printf("Small scale tests (on scene 'DoubleSlitSmallScale'):\n");
        using Scene = DoubleSlitSmallScale;

        // CPU.
        int32_t n_steps = Scene::t_end / Scene::dt;
        auto cpu_results = run_cpu<Scene>(0.0f, n_steps, 1, 1, wave_cpu<Scene>);
        writeBMP(
            "out/wave_cpu_small_scale.bmp",
            Scene::n_cells_x,
            Scene::n_cells_y,
            render_wave<Scene>(cpu_results.u0_final.data()));
        printf("  CPU sequential implementation:\n");
        printf("    run time: %.2f ms\n", cpu_results.time_ms);
        printf("\n");

        // GPU: wave_gpu_naive.
        auto gpu_naive_results =
            run_gpu_no_extra<Scene>(0.0f, n_steps, 1, 1, wave_gpu_naive<Scene>);
        writeBMP(
            "out/wave_gpu_naive_small_scale.bmp",
            Scene::n_cells_x,
            Scene::n_cells_y,
            render_wave<Scene>(gpu_naive_results.u0_final.data()));
        double naive_rel_rmse =
            rel_rmse(cpu_results.u0_final, gpu_naive_results.u0_final);
        if (naive_rel_rmse < tolerance) {
            gpu_naive_correct = true;
        }
        printf("  GPU naive implementation:\n");
        printf("    run time: %.2f ms\n", gpu_naive_results.time_ms);
        printf("    correctness: %.2e relative RMSE\n", naive_rel_rmse);
        printf("\n");

        // GPU: wave_gpu_shmem.
        auto gpu_shmem_results =
            run_gpu_extra<Scene>(0.0f, n_steps, 1, 1, wave_gpu_shmem<Scene>);
        writeBMP(
            "out/wave_gpu_shmem_small_scale.bmp",
            Scene::n_cells_x,
            Scene::n_cells_y,
            render_wave<Scene>(gpu_shmem_results.u0_final.data()));
        double shmem_rel_rmse =
            rel_rmse(cpu_results.u0_final, gpu_shmem_results.u0_final);
        if (shmem_rel_rmse < tolerance) {
            gpu_shmem_correct = true;
        }
        printf("  GPU shared memory implementation:\n");
        printf("    run time: %.2f ms\n", gpu_shmem_results.time_ms);
        printf("    correctness: %.2e relative RMSE\n", shmem_rel_rmse);
        printf("\n");

        if (gpu_naive_correct) {
            printf(
                "  CPU -> GPU naive speedup: %.2fx\n",
                cpu_results.time_ms / gpu_naive_results.time_ms);
        }
        if (gpu_shmem_correct) {
            printf(
                "  CPU -> GPU shared memory speedup: %.2fx\n",
                cpu_results.time_ms / gpu_shmem_results.time_ms);
        }
        if (gpu_naive_correct && gpu_shmem_correct) {
            printf(
                "  GPU naive -> GPU shared memory speedup: %.2fx\n",
                gpu_naive_results.time_ms / gpu_shmem_results.time_ms);
        }
        printf("\n");
    }

    // Run performance tests if requested.
    bool run_perf_tests = false;
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "-p") == 0) {
            run_perf_tests = true;
            break;
        }
    }

    // Large scale tests: mainly for performance.
    if (run_perf_tests) {
        printf("Large scale tests (on scene 'DoubleSlit'):\n");
        using Scene = DoubleSlit;

        int32_t n_steps = Scene::t_end / Scene::dt;
        int32_t num_iters_outer_gpu = 1;
        int32_t num_iters_inner_gpu = 1;

        // GPU: wave_gpu_naive.
        Results gpu_naive_results;
        if (gpu_naive_correct) {
            gpu_naive_results = run_gpu_no_extra<Scene>(
                0.0f,
                n_steps,
                num_iters_outer_gpu,
                num_iters_inner_gpu,
                wave_gpu_naive<Scene>);
            printf("  GPU naive implementation:\n");
            printf("    run time: %.2f ms\n", gpu_naive_results.time_ms);
            printf("\n");
            auto pixels_gpu_naive = render_wave<Scene>(gpu_naive_results.u0_final.data());
            writeBMP(
                "out/wave_gpu_naive_large_scale.bmp",
                Scene::n_cells_x,
                Scene::n_cells_y,
                pixels_gpu_naive);
        } else {
            printf("  Skipping GPU naive implementation (incorrect)\n");
        }

        // GPU: wave_gpu_shmem.
        Results gpu_shmem_results;
        double naive_shmem_rel_rmse = 0.0;
        if (gpu_shmem_correct) {
            gpu_shmem_results = run_gpu_extra<Scene>(
                0.0f,
                n_steps,
                num_iters_outer_gpu,
                num_iters_inner_gpu,
                wave_gpu_shmem<Scene>);
            naive_shmem_rel_rmse =
                rel_rmse(gpu_naive_results.u0_final, gpu_shmem_results.u0_final);
            printf("  GPU shared memory implementation:\n");
            printf("    run time: %.2f ms\n", gpu_shmem_results.time_ms);
            printf(
                "    correctness (w.r.t. GPU naive): %.2e relative RMSE\n",
                naive_shmem_rel_rmse);
            printf("\n");
            auto pixels_gpu_shmem = render_wave<Scene>(gpu_shmem_results.u0_final.data());
            writeBMP(
                "out/wave_gpu_shmem_large_scale.bmp",
                Scene::n_cells_x,
                Scene::n_cells_y,
                pixels_gpu_shmem);
        } else {
            printf("  Skipping GPU shared memory implementation (incorrect)\n");
        }

        if (gpu_naive_correct && gpu_shmem_correct && naive_shmem_rel_rmse < tolerance) {
            printf(
                "  GPU naive -> GPU shared memory speedup: %.2fx\n",
                gpu_naive_results.time_ms / gpu_shmem_results.time_ms);

        } else {
            printf("  GPU naive -> GPU shared memory speedup: N/A (incorrect)\n");
        }
        printf("\n");
    }

    // Generate animation if requested.
    bool a_flag = false;
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "-a") == 0) {
            a_flag = true;
            break;
        }
    }

    if (a_flag) {
        using Scene = DoubleSlitSmallScale;
        int32_t n_steps = Scene::t_end / Scene::dt;

        // CPU.
        FFmpegFrames cpu_frames;
        wave_ffmpeg<Scene>(0.0f, n_steps, cpu_frames);
        if (generate_animation<Scene>(cpu_frames, "out/wave_cpu") != 0) {
            std::cout << "CPU animation error: Failed to generate animation."
                      << std::endl;
        } else {
            std::cout << "CPU video has been generated." << std::endl;
        }

        // GPU naive.
        FFmpegFrames gpu_naive_frames;
        wave_ffmpeg_gpu<Scene>(0.0f, n_steps, gpu_naive_frames);
        if (generate_animation<Scene>(gpu_naive_frames, "out/wave_gpu_naive") != 0) {
            std::cout << "GPU_naive animation error: Failed to generate animation."
                      << std::endl;
        } else {
            std::cout << "GPU_naive video has been generated." << std::endl;
        }

        // GPU shared memory.
        FFmpegFrames gpu_shmem_frames;
        wave_ffmpeg_gpu_shmem<Scene>(0.0f, n_steps, gpu_shmem_frames);
        if (generate_animation<Scene>(gpu_shmem_frames, "out/wave_gpu_shmem") != 0) {
            std::cout << "GPU_shem animation error: Failed to generate animation."
                      << std::endl;
        } else {
            std::cout << "GPU_shmem video has been generated." << std::endl;
        }
    }

    return 0;
}
