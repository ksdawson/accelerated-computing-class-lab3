#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <limits>
#include <utility>
#include <vector>

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Already Written)

// 'wave_cpu_step':
//
// Input:
//
//     t -- time coordinate
//     u(t - dt) in array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t) in array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     u(t + dt) in array 'u0' (overwrites the input)
//
template <typename Scene> void wave_cpu_step(float t, float *u0, float const *u1) {
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;
    constexpr float c = Scene::c;
    constexpr float dx = Scene::dx;
    constexpr float dt = Scene::dt;

    for (int32_t idx_y = 0; idx_y < n_cells_y; ++idx_y) {
        for (int32_t idx_x = 0; idx_x < n_cells_x; ++idx_x) {
            int32_t idx = idx_y * n_cells_x + idx_x;
            bool is_border =
                (idx_x == 0 || idx_x == n_cells_x - 1 || idx_y == 0 ||
                 idx_y == n_cells_y - 1);
            float u_next_val;
            if (is_border || Scene::is_wall(idx_x, idx_y)) {
                u_next_val = 0.0f;
            } else if (Scene::is_source(idx_x, idx_y)) {
                u_next_val = Scene::source_value(idx_x, idx_y, t);
            } else {
                constexpr float coeff = c * c * dt * dt / (dx * dx);
                float damping = Scene::damping(idx_x, idx_y);
                u_next_val =
                    ((2.0f - damping - 4.0f * coeff) * u1[idx] -
                     (1.0f - damping) * u0[idx] +
                     coeff *
                         (u1[idx - 1] + u1[idx + 1] + u1[idx - n_cells_x] +
                          u1[idx + n_cells_x]));
            }
            u0[idx] = u_next_val;
        }
    }
}

// 'wave_cpu':
//
// Input:
//
//     t0 -- initial time coordinate
//     n_steps -- number of time steps to simulate
//     u(t0 - dt) in array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t0) in array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     Overwrites contents of memory pointed to by 'u0' and 'u1'
//
//     Returns pointers to buffers containing the final states of the wave
//     u(t0 + (n_steps - 1) * dt) and u(t0 + n_steps * dt).
//
template <typename Scene>
std::pair<float *, float *> wave_cpu(float t0, int32_t n_steps, float *u0, float *u1) {
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step++) {
        float t = t0 + idx_step * Scene::dt;
        wave_cpu_step<Scene>(t, u0, u1);
        std::swap(u0, u1);
    }
    return {u0, u1};
}

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// GPU Implementation (Naive)

// Helper to do wave math
template <typename Scene>
__device__ void wave(uint32_t idx_y, uint32_t idx_x, float t, // Scene params
    float *u0, float *u1, uint64_t memory_idx, uint32_t memory_height // Memory params
) {
    // Scene parameters
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;
    constexpr float c = Scene::c;
    constexpr float dx = Scene::dx;
    constexpr float dt = Scene::dt;

    // Wave math
    bool is_border =
        (idx_x == 0 || idx_x == n_cells_x - 1 || idx_y == 0 ||
            idx_y == n_cells_y - 1);
    float u_next_val;
    if (is_border || Scene::is_wall(idx_x, idx_y)) {
        u_next_val = 0.0f;
    } else if (Scene::is_source(idx_x, idx_y)) {
        u_next_val = Scene::source_value(idx_x, idx_y, t);
    } else {
        constexpr float coeff = c * c * dt * dt / (dx * dx);
        float damping = Scene::damping(idx_x, idx_y);
        u_next_val =
            ((2.0f - damping - 4.0f * coeff) * u1[memory_idx] -
                (1.0f - damping) * u0[memory_idx] +
                coeff *
                    (u1[memory_idx - 1] + u1[memory_idx + 1] + u1[memory_idx - memory_height] +
                    u1[memory_idx + memory_height]));
    }
    u0[memory_idx] = u_next_val;
}

// 'wave_gpu_step':
//
// Input:
//
//     t -- time coordinate
//     u(t - dt) in GPU array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t) in GPU array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     u(t + dt) in GPU array 'u0' (overwrites the input)
//
template <typename Scene>
__global__ void wave_gpu_naive_step(
    float t,
    float *u0, /* pointer to GPU memory */
    float *u1, /* pointer to GPU memory */
    uint8_t ilp_size = 1
) {
    // Scene parameters
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;

    // Thread info
    int tot_threads = gridDim.x * blockDim.x;
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Flatten the 2D iteration space into 1D and stride tot_threads pixels each iteration
    for (uint64_t idx = thread_index * ilp_size; idx < n_cells_y * n_cells_x; idx += tot_threads * ilp_size) {
        #pragma unroll
        for (uint8_t i = 0; i < ilp_size; ++i) {
            // Use 32x1 vectors
            uint64_t ilp_idx = idx + i;
            uint32_t idx_y = ilp_idx / n_cells_x;
            uint32_t idx_x = ilp_idx % n_cells_x;
            // Wave math
            wave<Scene>(idx_y, idx_x, t,
                u0, u1, ilp_idx, n_cells_x
            );
        }
    }
}

// 'wave_gpu_naive':
//
// Input:
//
//     t0 -- initial time coordinate
//     n_steps -- number of time steps to simulate
//     u(t0 - dt) in GPU array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t0) in GPU array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     Launches kernels to overwrite the GPU memory pointed to by 'u0' and 'u1'
//
//     Returns pointers to GPU buffers which will contain the final states of
//     the wave u(t0 + (n_steps - 1) * dt) and u(t0 + n_steps * dt) after all
//     launched kernels have completed.
//
template <typename Scene>
std::pair<float *, float *> wave_gpu_naive(
    float t0,
    int32_t n_steps,
    float *u0, /* pointer to GPU memory */
    float *u1  /* pointer to GPU memory */
) {
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step++) {
        float t = t0 + idx_step * Scene::dt;
        wave_gpu_naive_step<Scene><<<48, 32 * 32>>>(t, u0, u1);
        std::swap(u0, u1);
    }
    return {u0, u1};
}

////////////////////////////////////////////////////////////////////////////////
// GPU Implementation (With Shared Memory)

// Helpers to load/store data
__device__ void copy_mem(
    float *src_u0, float *src_u1, uint32_t src_buffer_height,
    float *dst_u0, float *dst_u1, uint32_t dst_buffer_height,
    uint32_t local_height, uint32_t local_width
) {
    // Copy data from one buffer to another
    for (uint64_t local_idx = threadIdx.x; local_idx < local_height * local_width; local_idx += blockDim.x) {
        // Get local idx for local offset
        uint32_t local_idx_y = local_idx / local_height;
        uint32_t local_idx_x = local_idx % local_height;
        // Get src idx
        uint32_t src_idx = local_idx_y * src_buffer_height + local_idx_x;
        // Get dst idx
        uint32_t dst_idx = local_idx_y * dst_buffer_height + local_idx_x;
        // Copy memory over
        dst_u0[dst_idx] = src_u0[src_idx];
        dst_u1[dst_idx] = src_u1[src_idx];
    }
    // Wait for all the memory to be copied
    __syncthreads();
}
__device__ void load_shmem(
    float *global_u0, float *global_u1, uint32_t global_buffer_height, // Main memory buffer params
    float *local_u0, float *local_u1, uint32_t local_buffer_height, // SRAM buffer params
    uint32_t tile_height, uint32_t tile_width // Tile params
) {
    // Load data from main memory to SRAM
    copy_mem(global_u0, global_u1, global_buffer_height,
        local_u0, local_u1, local_buffer_height,
        tile_height, tile_width
    );
}
__device__ void store_shmem(
    float *global_u0, float *global_u1, uint32_t global_buffer_height, // Main memory buffer params
    float *local_u0, float *local_u1, uint32_t local_buffer_height, // SRAM buffer params
    uint32_t tile_height, uint32_t tile_width // Tile params
) {
    // Store data to main memory from SRAM
    copy_mem(local_u0, local_u1, local_buffer_height,
        global_u0, global_u1, global_buffer_height,
        tile_height, tile_width
    );
}

// Helper to setup the tile
__device__ void setup_tile(uint32_t scene_height, uint32_t scene_width, uint8_t tile_padding, // Input
    uint32_t tiles_per_col, uint32_t tiles_per_row, uint32_t tile_j, uint32_t tile_i,
    uint32_t *out_tile_height, uint32_t *out_tile_width, uint32_t *out_scene_idx, // Output
    uint8_t *left_shrink_amt, uint8_t *right_shrink_amt, uint8_t *top_shrink_amt, uint8_t *bottom_shrink_amt // Output
) {
    // Divide the scene into tiles (valid data that must be written back at the end)
    uint32_t tile_height = scene_height / tiles_per_col;
    uint32_t tile_width = scene_width / tiles_per_row;

    // Calculate starting scene idx of the tile
    uint32_t scene_idx_y = tile_j * tile_width;
    uint32_t scene_idx_x = tile_i * tile_height;

    // Handle grids not divisible by the number of SMs
    uint8_t extra_rows = scene_height % tiles_per_col;
    uint8_t extra_cols = scene_width % tiles_per_row;
    // Assign the extra to the edges since they have smaller overlap (limit to last for simplicity)
    tile_width += (tile_j == tiles_per_row - 1) ? extra_cols : 0;
    tile_height += (tile_i == tiles_per_col - 1) ? extra_rows : 0;

    // You can only expand to the edges
    uint8_t tile_padding_left = min(scene_idx_y, tile_padding);
    uint8_t tile_padding_right = min(scene_width - (scene_idx_y + tile_width), tile_padding);
    uint8_t tile_padding_top = min(scene_idx_x, tile_padding);
    uint8_t tile_padding_bottom = min(scene_height - (scene_idx_x + tile_height), tile_padding);

    // Expand the tile by the number of time steps in each direction (overlap for invalid data)
    tile_width += tile_padding_left + tile_padding_right;
    tile_height += tile_padding_top + tile_padding_bottom;

    // Update the scene idx based on the expansion
    scene_idx_y -= tile_padding_left;
    scene_idx_x -= tile_padding_top;
    uint32_t scene_idx = scene_idx_y * scene_height + scene_idx_x;

    // Set the tile
    *out_tile_height = tile_height;
    *out_tile_width = tile_width;
    *out_scene_idx = scene_idx;

    // Set the shrink amts (edges can only shrink in one dir)
    // If your expansion hit an edge, then you should only shrink when the opposite shrinks are equal to ensure you shrink to the correct size
    *left_shrink_amt = tile_padding_left;
    *right_shrink_amt = tile_padding_right;
    *top_shrink_amt = tile_padding_top;
    *bottom_shrink_amt = tile_padding_bottom;
}

// Helper to shrink along one dimension
__device__ void shrink_axis(uint32_t &tile_size, uint8_t &front_shrink, uint8_t &back_shrink, // Tile params
    float* &global_u0, float* &global_u1, uint32_t global_stride, // Global buffer params
    float* &local_u0, float* &local_u1, uint32_t local_stride // Local buffer params
) {
    // Move the buffers
    if (front_shrink >= back_shrink) {
        global_u0 += global_stride;
        global_u1 += global_stride;
        local_u0  += local_stride;
        local_u1  += local_stride;
    }
    // Shrink the tile size
    if (front_shrink > back_shrink) {
        --tile_size;
        --front_shrink;
    } else if (front_shrink < back_shrink) {
        --tile_size;
        --back_shrink;
    } else {
        tile_size -= 2;
        --front_shrink;
        --back_shrink;
    }
}

template <typename Scene>
__global__ void wave_gpu_shmem_multistep(
    float t0, uint32_t ti_step, uint32_t tf_step, // Time params
    float *u0, float *u1, float *extra0, float *extra1, // Buffer params
    uint32_t subtiles_per_col, uint32_t subtiles_per_row // Loop tiling params
) {
    // Setup the block's SRAM
    extern __shared__ float sram[];

    // Iterate over the SM's subtiles and process each one one at a time
    for (uint32_t subtile_idx = blockIdx.x; subtile_idx < subtiles_per_col * subtiles_per_row; subtile_idx += gridDim.x) {
        // Get subtile idx
        uint32_t subtile_idx_y = subtile_idx / subtiles_per_col;
        uint32_t subtile_idx_x = subtile_idx % subtiles_per_col;

        // Global buffers
        float *global_u0 = u0;
        float *global_u1 = u1;
        uint32_t global_buffer_height = Scene::n_cells_x;

        // Tile dimensions
        uint32_t tile_height, tile_width;
        // Shrink parameters
        uint8_t left_shrink_amt, right_shrink_amt, top_shrink_amt, bottom_shrink_amt;
        // Setup the tile
        uint8_t tile_padding = tf_step - ti_step; // Expand by the number of time steps
        uint32_t scene_idx;
        setup_tile(Scene::n_cells_x, Scene::n_cells_y, tile_padding,
            subtiles_per_col, subtiles_per_row, subtile_idx_y, subtile_idx_x,
            &tile_height, &tile_width, &scene_idx,
            &left_shrink_amt, &right_shrink_amt, &top_shrink_amt, &bottom_shrink_amt
        );
        // Move global buffers to tile location
        global_u0 += scene_idx;
        global_u1 += scene_idx;

        // Local buffers
        float *local_u0 = sram;
        float *local_u1 = sram + tile_height * tile_width;
        uint32_t local_buffer_height = tile_height;
        
        // Load data from main (u0, u1) to local memory
        load_shmem(global_u0, global_u1, global_buffer_height,
            local_u0, local_u1, local_buffer_height,
            tile_height, tile_width
        );

        // Limit steps so we don’t exceed shrink
        uint8_t max_shrink = max(
            max(left_shrink_amt, right_shrink_amt),
            max(top_shrink_amt, bottom_shrink_amt)
        );
        uint32_t actual_tf_step = ti_step + min(tf_step - ti_step, max_shrink);

        // Iterate over the time steps
        for (uint32_t idx_step = ti_step; idx_step < actual_tf_step; ++idx_step) {
            // Shrink tile width
            shrink_axis(tile_width, left_shrink_amt, right_shrink_amt,
                global_u0, global_u1, global_buffer_height,
                local_u0, local_u1, local_buffer_height
            );
            // Shrink tile height
            shrink_axis(tile_height, top_shrink_amt, bottom_shrink_amt,
                global_u0, global_u1, 1, 
                local_u0, local_u1, 1
            );

            // Calculate t
            float t = t0 + idx_step * Scene::dt;

            // Flatten the 2D iteration space into 1D and stride tot_threads pixels each iteration
            for (uint64_t tile_idx = threadIdx.x; tile_idx < tile_height * tile_width; tile_idx += blockDim.x) {
                // Get tile idx for tile offset
                uint32_t tile_idx_y = tile_idx / tile_height;
                uint32_t tile_idx_x = tile_idx % tile_height;
                // Get global idx for calculation
                uint64_t global_idx = global_u0 - u0;
                global_idx += tile_idx_y * global_buffer_height + tile_idx_x;
                uint32_t global_idx_y = global_idx / global_buffer_height;
                uint32_t global_idx_x = global_idx % global_buffer_height;
                // Get local idx for memory
                uint64_t local_idx = tile_idx_y * local_buffer_height + tile_idx_x;
                // Wave math
                wave<Scene>(global_idx_y, global_idx_x, t,
                    local_u0, local_u1,
                    local_idx, local_buffer_height
                );
            }

            // We need the new pixel for all pixels in the block before processing the next time step
            __syncthreads();

            // Swap the local buffer pointers
            std::swap(local_u0, local_u1);
        }

        // Swap the local buffer pointers back
        std::swap(local_u0, local_u1);

        // We need to use the extra buffers for storing to keep main memory immutable while a block processes multiple tiles
        float *extra_global_u0 = (global_u0 - u0) + extra0;
        float *extra_global_u1 = (global_u1 - u1) + extra1;
        // Store data from local memory to main memory (extra0, extra1)
        store_shmem(extra_global_u0, extra_global_u1, global_buffer_height,
            local_u0, local_u1, local_buffer_height,
            tile_height, tile_width
        );
    }
}

// 'wave_gpu_shmem':
//
// Input:
//
//     t0 -- initial time coordinate
//
//     n_steps -- number of time steps to simulate
//
//     u(t0 - dt) in GPU array 'u0' of size 'n_cells_y * n_cells_x'
///
//     u(t0) in GPU array 'u1' of size 'n_cells_y * n_cells_x'
//
//     Scratch buffers 'extra0' and 'extra1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     Launches kernels to (potentially) overwrite the GPU memory pointed to
//     by 'u0' and 'u1', 'extra0', and 'extra1'.
//
//     Returns pointers to GPU buffers which will contain the final states of
//     the wave u(t0 + (n_steps - 1) * dt) and u(t0 + n_steps * dt) after all
//     launched kernels have completed. These buffers can be any of 'u0', 'u1',
//     'extra0', or 'extra1'.
//
template <typename Scene>
std::pair<float *, float *> wave_gpu_shmem(
    float t0,
    int32_t n_steps,
    float *u0,     /* pointer to GPU memory */
    float *u1,     /* pointer to GPU memory */
    float *extra0, /* pointer to GPU memory */
    float *extra1  /* pointer to GPU memory */
) {
    // Number of time steps to process at once in a kernel
    bool large_scene = Scene::n_cells_y * Scene::n_cells_x > 600000; // Could be useful?
    uint8_t time_steps = large_scene ? 8 : 8;

    // Assume we want tw = th for square tiles then
    // Min subtile size math:
    // (1) tw = th = w/tpr = h/tpc
    // (2) tpr * tpc = 48 => tpr = 48/tpc
    // (3) w/(48/tpc) = h/tpc => tpc = sqrt(48*h/w) 
    // (4) tw = th = h/sqrt(48*h/w) => tw = th = sqrt(h)*sqrt(w)/sqrt(48)
    // Max subtile size math:
    // (1) 2 * (tw + 2t) * (th + 2t) * ((32/8)/1000) <= MAX_SRAM (100)
    // (2) (tw + 2t) * (tw + 2t) <= 12500
    // (3) tw <= sqrt(6250) - 2t
    // (4) To maximize SRAM we will use tw = sqrt(6250) - 2t
    uint16_t subtile_width = min(
        (uint64_t)(sqrt(Scene::n_cells_y) * sqrt(Scene::n_cells_x) / sqrt(48)),
        (uint64_t)(sqrt(6250) - 2 * time_steps)
    );
    uint16_t subtile_height = subtile_width;

    // Subtile dimensions
    uint32_t subtiles_per_row = Scene::n_cells_y / subtile_width;
    uint32_t subtiles_per_col = Scene::n_cells_x / subtile_height;
    if (subtiles_per_row * subtiles_per_col < 48) {
        // Add the extra tiles to the height
        subtiles_per_col += (48 - subtiles_per_row * subtiles_per_col) / subtiles_per_row;
    }

    for (uint32_t idx_step = 0; idx_step < n_steps; idx_step += time_steps) {
        // Compute starting and ending time step
        uint32_t ti_step = idx_step;
        uint32_t tf_step = ti_step + min(n_steps - idx_step, time_steps);

        // Setup the block SRAM
        int shmem_size_bytes = 100 * 1000; // Max 100 KB per block
        CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
            wave_gpu_shmem_multistep<Scene>),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            shmem_size_bytes
        ));

        // Launch our kernel
        wave_gpu_shmem_multistep<Scene><<<48, 32 * 32, shmem_size_bytes>>>(t0, ti_step, tf_step, u0, u1, extra0, extra1, subtiles_per_col, subtiles_per_row);

        // Treat the multi step kernel as one step so u0 will now contain the most recent
        std::swap(u0, extra0);
        std::swap(u1, extra1);
        std::swap(u0, u1);
    }
    return {u0, u1};
}

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

struct BaseScene {
    constexpr static int32_t n_cells_x = 3201;
    constexpr static int32_t n_cells_y = 3201;
    constexpr static float c = 1.0f;
    constexpr static float dx = 1.0f / float(n_cells_x - 1);
    constexpr static float dy = 1.0f / float(n_cells_y - 1);
    constexpr static float dt = 0.25f * dx / c;
    constexpr static float t_end = 1.0f;
};

struct BaseSceneSmallScale {
    constexpr static int32_t n_cells_x = 201;
    constexpr static int32_t n_cells_y = 201;
    constexpr static float c = 1.0f;
    constexpr static float dx = 1.0f / float(n_cells_x - 1);
    constexpr static float dy = 1.0f / float(n_cells_y - 1);
    constexpr static float dt = 0.25f * dx / c;
    constexpr static float t_end = 1.0f;
};

float __host__ __device__ __forceinline__ boundary_damping(
    int32_t n_cells_x,
    int32_t n_cells_y,
    float ramp_size,
    float max_damping,
    int32_t idx_x,
    int32_t idx_y) {
    float x = float(idx_x) / (n_cells_x - 1);
    float y = float(idx_y) / (n_cells_y - 1);
    float fx = 1.0f - min(min(x, 1.0f - x), ramp_size) / ramp_size;
    float fy = 1.0f - min(min(y, 1.0f - y), ramp_size) / ramp_size;
    float f = max(fx, fy);
    return max_damping * f * f;
}

struct PointSource : public BaseScene {
    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        return false;
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 2 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 20.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

struct Slit : public BaseScene {
    constexpr static float slit_width = 0.05f;

    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        float y = float(idx_y) / (n_cells_y - 1);
        return idx_x == (n_cells_x - 1) / 2 &&
            (y < 0.5f - slit_width / 2 || y > 0.5f + slit_width / 2);
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 4 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 40.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

struct DoubleSlit : public BaseScene {
    constexpr static float slit_width = 0.03f;

    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        float y = float(idx_y) / (n_cells_y - 1);
        return (idx_x == (n_cells_x - 1) * 2 / 3) &&
            !((y >= 0.45f - slit_width / 2 && y <= 0.45f + slit_width / 2) ||
              (y >= 0.55f - slit_width / 2 && y <= 0.55f + slit_width / 2));
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 6 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 20.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

struct DoubleSlitSmallScale : public BaseSceneSmallScale {
    constexpr static float slit_width = 0.03f;

    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        constexpr float EPS = 1e-6;
        float y = float(idx_y) / (n_cells_y - 1);
        return (idx_x == (n_cells_x - 1) * 2 / 3) &&
            !((y >= 0.45f - slit_width / 2 - EPS && y <= 0.45f + slit_width / 2 + EPS) ||
              (y >= 0.55f - slit_width / 2 - EPS && y <= 0.55f + slit_width / 2 + EPS));
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 6 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 20.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

// Output image writers: BMP file header structure
#pragma pack(push, 1)
struct BMPHeader {
    uint16_t fileType{0x4D42};   // File type, always "BM"
    uint32_t fileSize{0};        // Size of the file in bytes
    uint16_t reserved1{0};       // Always 0
    uint16_t reserved2{0};       // Always 0
    uint32_t dataOffset{54};     // Start position of pixel data
    uint32_t headerSize{40};     // Size of this header (40 bytes)
    int32_t width{0};            // Image width in pixels
    int32_t height{0};           // Image height in pixels
    uint16_t planes{1};          // Number of color planes
    uint16_t bitsPerPixel{24};   // Bits per pixel (24 for RGB)
    uint32_t compression{0};     // Compression method (0 for uncompressed)
    uint32_t imageSize{0};       // Size of raw bitmap data
    int32_t xPixelsPerMeter{0};  // Horizontal resolution
    int32_t yPixelsPerMeter{0};  // Vertical resolution
    uint32_t colorsUsed{0};      // Number of colors in the color palette
    uint32_t importantColors{0}; // Number of important colors
};
#pragma pack(pop)

void writeBMP(
    const char *fname,
    uint32_t width,
    uint32_t height,
    const std::vector<uint8_t> &pixels) {
    BMPHeader header;
    header.width = width;
    header.height = height;

    uint32_t rowSize = (width * 3 + 3) & (~3); // Align to 4 bytes
    header.imageSize = rowSize * height;
    header.fileSize = header.dataOffset + header.imageSize;

    std::ofstream file(fname, std::ios::binary);
    file.write(reinterpret_cast<const char *>(&header), sizeof(header));

    // Write pixel data with padding
    std::vector<uint8_t> padding(rowSize - width * 3, 0);
    for (int32_t idx_y = height - 1; idx_y >= 0;
         --idx_y) { // BMP stores pixels from bottom to top
        const uint8_t *row = &pixels[idx_y * width * 3];
        file.write(reinterpret_cast<const char *>(row), width * 3);
        if (!padding.empty()) {
            file.write(reinterpret_cast<const char *>(padding.data()), padding.size());
        }
    }
}

// If trunc - cut the border of the image.
template <typename Scene>
std::vector<uint8_t> render_wave(const float *u, int trunc = 0) {
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;

    std::vector<uint8_t> pixels((n_cells_x - trunc) * (n_cells_y - trunc) * 3);
    for (int32_t idx_y = 0; idx_y < n_cells_y - trunc; ++idx_y) {
        for (int32_t idx_x = 0; idx_x < n_cells_x - trunc; ++idx_x) {
            int32_t idx = idx_y * (n_cells_x - trunc) + idx_x;
            int32_t u_idx = idx_y * n_cells_x + idx_x;
            float val = u[u_idx];
            bool is_wall = Scene::is_wall(idx_x, idx_y);
            // BMP stores pixels in BGR order
            if (is_wall) {
                pixels[idx * 3 + 2] = 0;
                pixels[idx * 3 + 1] = 0;
                pixels[idx * 3 + 0] = 0;
            } else if (val > 0.0f) {
                pixels[idx * 3 + 2] = 255;
                pixels[idx * 3 + 1] = 255 - uint8_t(min(val * 255.0f, 255.0f));
                pixels[idx * 3 + 0] = 255 - uint8_t(min(val * 255.0f, 255.0f));
            } else {
                pixels[idx * 3 + 2] = 255 - uint8_t(min(-val * 255.0f, 255.0f));
                pixels[idx * 3 + 1] = 255 - uint8_t(min(-val * 255.0f, 255.0f));
                pixels[idx * 3 + 0] = 255;
            }
        }
    }
    return pixels;
}

struct Results {
    std::vector<float> u0_final;
    std::vector<float> u1_final;
    double time_ms;
};

template <typename Scene, typename F>
Results run_cpu(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    F func) {
    auto u0 = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u1 = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);

    std::pair<float *, float *> u_final;

    double best_time = std::numeric_limits<double>::infinity();
    for (int32_t i = 0; i < num_iters_outer; ++i) {
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t j = 0; j < num_iters_inner; ++j) {
            std::fill(u0.begin(), u0.end(), 0.0f);
            std::fill(u1.begin(), u1.end(), 0.0f);
            u_final = func(t0, n_steps, u0.data(), u1.data());
        }
        auto end = std::chrono::high_resolution_clock::now();
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count() /
            num_iters_inner;
        best_time = std::min(best_time, time_ms);
    }

    if (u_final.first == u1.data() && u_final.second == u0.data()) {
        std::swap(u0, u1);
    } else if (!(u_final.first == u0.data() && u_final.second == u1.data())) {
        std::cerr << "Unexpected return values from 'func'" << std::endl;
        std::abort();
    }

    return {std::move(u0), std::move(u1), best_time};
}

template <typename Scene, typename F>
Results run_gpu(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    bool use_extra,
    F func) {
    float *u0;
    float *u1;
    float *extra0 = nullptr;
    float *extra1 = nullptr;

    CUDA_CHECK(hipMalloc(&u0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&u1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));

    if (use_extra) {
        CUDA_CHECK(
            hipMalloc(&extra0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
        CUDA_CHECK(
            hipMalloc(&extra1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    }

    std::pair<float *, float *> u_final;

    double best_time = std::numeric_limits<double>::infinity();
    for (int32_t i = 0; i < num_iters_outer; ++i) {
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t j = 0; j < num_iters_inner; ++j) {
            CUDA_CHECK(
                hipMemset(u0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
            CUDA_CHECK(
                hipMemset(u1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
            if (use_extra) {
                CUDA_CHECK(hipMemset(
                    extra0,
                    0,
                    Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
                CUDA_CHECK(hipMemset(
                    extra1,
                    0,
                    Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
            }
            u_final = func(t0, n_steps, u0, u1, extra0, extra1);
        }
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count() /
            num_iters_inner;
        best_time = std::min(best_time, time_ms);
    }

    if (u_final.first != u0 && u_final.first != u1 &&
        (extra0 == nullptr || u_final.first != extra0) &&
        (extra1 == nullptr || u_final.first != extra1)) {
        std::cerr << "Unexpected final 'u0' pointer returned from GPU implementation"
                  << std::endl;
        std::abort();
    }

    if (u_final.second != u0 && u_final.second != u1 &&
        (extra0 == nullptr || u_final.second != extra0) &&
        (extra1 == nullptr || u_final.second != extra1)) {
        std::cerr << "Unexpected final 'u1' pointer returned from GPU implementation"
                  << std::endl;
        std::abort();
    }

    auto u0_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u1_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    CUDA_CHECK(hipMemcpy(
        u0_cpu.data(),
        u_final.first,
        Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
        hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(
        u1_cpu.data(),
        u_final.second,
        Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
        hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(u0));
    CUDA_CHECK(hipFree(u1));
    if (use_extra) {
        CUDA_CHECK(hipFree(extra0));
        CUDA_CHECK(hipFree(extra1));
    }

    return {std::move(u0_cpu), std::move(u1_cpu), best_time};
}

template <typename Scene, typename F>
Results run_gpu_extra(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    F func) {
    return run_gpu<Scene>(t0, n_steps, num_iters_outer, num_iters_inner, true, func);
}

template <typename Scene, typename F>
Results run_gpu_no_extra(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    F func) {
    return run_gpu<Scene>(
        t0,
        n_steps,
        num_iters_outer,
        num_iters_inner,
        false,
        [&](float t0,
            int32_t n_steps,
            float *u0,
            float *u1,
            float *extra0,
            float *extra1) { return func(t0, n_steps, u0, u1); });
}

double rel_rmse(std::vector<float> const &a, std::vector<float> const &b) {
    if (a.size() != b.size()) {
        std::cerr << "Mismatched sizes in 'rel_rmse'" << std::endl;
        std::abort();
    }
    double ref_sum = 0.0;
    double sum = 0.0;
    for (size_t i = 0; i < a.size(); ++i) {
        ref_sum += double(a.at(i)) * double(a.at(i));
        double diff = double(a.at(i)) - double(b.at(i));
        sum += diff * diff;
    }
    return sqrt(sum / a.size()) / sqrt(ref_sum / a.size());
}

// FFmpeg implementations.
typedef std::vector<std::vector<uint8_t>> FFmpegFrames;

// CPU implementation with FFmpeg framing.
template <typename Scene>
void wave_ffmpeg(float t0, int32_t n_steps, FFmpegFrames &frames) {
    static constexpr int32_t frame_step = 2;
    auto u0_v = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u1_v = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u0 = u0_v.data();
    auto u1 = u1_v.data();
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step += frame_step) {
        auto r = wave_cpu<Scene>(t0 + idx_step * Scene::dt, frame_step, u0, u1);
        u0 = r.first;
        u1 = r.second;
        frames.push_back(render_wave<Scene>(u1, 1));
    }
}

template <typename Scene>
void wave_ffmpeg_gpu(float t0, int32_t n_steps, FFmpegFrames &frames) {
    static constexpr int32_t frame_step = 2;
    auto u1_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    float *u0;
    float *u1;
    CUDA_CHECK(hipMalloc(&u0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&u1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step += frame_step) {
        auto r = wave_gpu_naive<Scene>(t0 + idx_step * Scene::dt, frame_step, u0, u1);
        u0 = r.first;
        u1 = r.second;
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(
            u1_cpu.data(),
            u1,
            Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
            hipMemcpyDeviceToHost));
        frames.push_back(render_wave<Scene>(u1_cpu.data(), 1));
    }
}

template <typename Scene>
void wave_ffmpeg_gpu_shmem(float t0, int32_t n_steps, FFmpegFrames &frames) {
    static constexpr int32_t frame_step = 2;
    auto u1_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    float *u0;
    float *u1;
    float *extra0;
    float *extra1;
    CUDA_CHECK(hipMalloc(&u0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&u1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&extra0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&extra1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    float *buffers[] = {u0, u1, extra0, extra1};
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step += frame_step) {
        CUDA_CHECK(
            hipMemset(extra0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
        CUDA_CHECK(
            hipMemset(extra1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
        auto r = wave_gpu_shmem<Scene>(
            t0 + idx_step * Scene::dt,
            frame_step,
            u0,
            u1,
            extra0,
            extra1);
        u0 = r.first;
        u1 = r.second;
        for (int i = 0; i < 4; ++i) {
            if (buffers[i] != u0 && buffers[i] != u1) {
                extra0 = buffers[i];
            }
        }
        for (int i = 0; i < 4; ++i) {
            if (buffers[i] != u0 && buffers[i] != u1 && buffers[i] != extra0) {
                extra1 = buffers[i];
            }
        }
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(
            u1_cpu.data(),
            u1,
            Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
            hipMemcpyDeviceToHost));
        frames.push_back(render_wave<Scene>(u1_cpu.data(), 1));
    }
}

template <typename Scene>
int generate_animation(const FFmpegFrames &frames, std::string fname) {
    std::string ffmpeg_command = "ffmpeg -y "
                                 "-f rawvideo "
                                 "-pixel_format rgb24 "
                                 "-video_size " +
        std::to_string(Scene::n_cells_x - 1) + "x" +
        std::to_string(Scene::n_cells_y - 1) +
        " "
        "-framerate " +
        std::to_string(30) +
        " "
        "-i - "
        "-c:v libx264 "
        "-pix_fmt yuv420p " +
        fname + ".mp4" + " 2> /dev/null";

    FILE *pipe = popen(ffmpeg_command.c_str(), "w");
    if (!pipe) {
        std::cerr << "Failed to open pipe to FFmpeg." << std::endl;
        return 1;
    }

    for (auto &frame : frames) {
        if (fwrite(frame.data(), 1, frame.size(), pipe) != frame.size()) {
            std::cerr << "Failed to write frame to FFmpeg." << std::endl;
            return 1;
        }
    }

    pclose(pipe);
    return 0;
}

int main(int argc, char **argv) {
    // Small scale tests: mainly for correctness.
    double tolerance = 3e-2;
    bool gpu_naive_correct = false;
    bool gpu_shmem_correct = false;
    {
        printf("Small scale tests (on scene 'DoubleSlitSmallScale'):\n");
        using Scene = DoubleSlitSmallScale;

        // CPU.
        int32_t n_steps = Scene::t_end / Scene::dt;
        auto cpu_results = run_cpu<Scene>(0.0f, n_steps, 1, 1, wave_cpu<Scene>);
        writeBMP(
            "out/wave_cpu_small_scale.bmp",
            Scene::n_cells_x,
            Scene::n_cells_y,
            render_wave<Scene>(cpu_results.u0_final.data()));
        printf("  CPU sequential implementation:\n");
        printf("    run time: %.2f ms\n", cpu_results.time_ms);
        printf("\n");

        // GPU: wave_gpu_naive.
        auto gpu_naive_results =
            run_gpu_no_extra<Scene>(0.0f, n_steps, 1, 1, wave_gpu_naive<Scene>);
        writeBMP(
            "out/wave_gpu_naive_small_scale.bmp",
            Scene::n_cells_x,
            Scene::n_cells_y,
            render_wave<Scene>(gpu_naive_results.u0_final.data()));
        double naive_rel_rmse =
            rel_rmse(cpu_results.u0_final, gpu_naive_results.u0_final);
        if (naive_rel_rmse < tolerance) {
            gpu_naive_correct = true;
        }
        printf("  GPU naive implementation:\n");
        printf("    run time: %.2f ms\n", gpu_naive_results.time_ms);
        printf("    correctness: %.2e relative RMSE\n", naive_rel_rmse);
        printf("\n");

        // GPU: wave_gpu_shmem.
        auto gpu_shmem_results =
            run_gpu_extra<Scene>(0.0f, n_steps, 1, 1, wave_gpu_shmem<Scene>);
        writeBMP(
            "out/wave_gpu_shmem_small_scale.bmp",
            Scene::n_cells_x,
            Scene::n_cells_y,
            render_wave<Scene>(gpu_shmem_results.u0_final.data()));
        double shmem_rel_rmse =
            rel_rmse(cpu_results.u0_final, gpu_shmem_results.u0_final);
        if (shmem_rel_rmse < tolerance) {
            gpu_shmem_correct = true;
        }
        printf("  GPU shared memory implementation:\n");
        printf("    run time: %.2f ms\n", gpu_shmem_results.time_ms);
        printf("    correctness: %.2e relative RMSE\n", shmem_rel_rmse);
        printf("\n");

        if (gpu_naive_correct) {
            printf(
                "  CPU -> GPU naive speedup: %.2fx\n",
                cpu_results.time_ms / gpu_naive_results.time_ms);
        }
        if (gpu_shmem_correct) {
            printf(
                "  CPU -> GPU shared memory speedup: %.2fx\n",
                cpu_results.time_ms / gpu_shmem_results.time_ms);
        }
        if (gpu_naive_correct && gpu_shmem_correct) {
            printf(
                "  GPU naive -> GPU shared memory speedup: %.2fx\n",
                gpu_naive_results.time_ms / gpu_shmem_results.time_ms);
        }
        printf("\n");
    }

    // Run performance tests if requested.
    bool run_perf_tests = false;
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "-p") == 0) {
            run_perf_tests = true;
            break;
        }
    }

    // Large scale tests: mainly for performance.
    if (run_perf_tests) {
        printf("Large scale tests (on scene 'DoubleSlit'):\n");
        using Scene = DoubleSlit;

        int32_t n_steps = Scene::t_end / Scene::dt;
        int32_t num_iters_outer_gpu = 1;
        int32_t num_iters_inner_gpu = 1;

        // GPU: wave_gpu_naive.
        Results gpu_naive_results;
        if (gpu_naive_correct) {
            gpu_naive_results = run_gpu_no_extra<Scene>(
                0.0f,
                n_steps,
                num_iters_outer_gpu,
                num_iters_inner_gpu,
                wave_gpu_naive<Scene>);
            printf("  GPU naive implementation:\n");
            printf("    run time: %.2f ms\n", gpu_naive_results.time_ms);
            printf("\n");
            auto pixels_gpu_naive = render_wave<Scene>(gpu_naive_results.u0_final.data());
            writeBMP(
                "out/wave_gpu_naive_large_scale.bmp",
                Scene::n_cells_x,
                Scene::n_cells_y,
                pixels_gpu_naive);
        } else {
            printf("  Skipping GPU naive implementation (incorrect)\n");
        }

        // GPU: wave_gpu_shmem.
        Results gpu_shmem_results;
        double naive_shmem_rel_rmse = 0.0;
        if (gpu_shmem_correct) {
            gpu_shmem_results = run_gpu_extra<Scene>(
                0.0f,
                n_steps,
                num_iters_outer_gpu,
                num_iters_inner_gpu,
                wave_gpu_shmem<Scene>);
            naive_shmem_rel_rmse =
                rel_rmse(gpu_naive_results.u0_final, gpu_shmem_results.u0_final);
            printf("  GPU shared memory implementation:\n");
            printf("    run time: %.2f ms\n", gpu_shmem_results.time_ms);
            printf(
                "    correctness (w.r.t. GPU naive): %.2e relative RMSE\n",
                naive_shmem_rel_rmse);
            printf("\n");
            auto pixels_gpu_shmem = render_wave<Scene>(gpu_shmem_results.u0_final.data());
            writeBMP(
                "out/wave_gpu_shmem_large_scale.bmp",
                Scene::n_cells_x,
                Scene::n_cells_y,
                pixels_gpu_shmem);
        } else {
            printf("  Skipping GPU shared memory implementation (incorrect)\n");
        }

        if (gpu_naive_correct && gpu_shmem_correct && naive_shmem_rel_rmse < tolerance) {
            printf(
                "  GPU naive -> GPU shared memory speedup: %.2fx\n",
                gpu_naive_results.time_ms / gpu_shmem_results.time_ms);

        } else {
            printf("  GPU naive -> GPU shared memory speedup: N/A (incorrect)\n");
        }
        printf("\n");
    }

    // Generate animation if requested.
    bool a_flag = false;
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "-a") == 0) {
            a_flag = true;
            break;
        }
    }

    if (a_flag) {
        using Scene = DoubleSlitSmallScale;
        int32_t n_steps = Scene::t_end / Scene::dt;

        // CPU.
        FFmpegFrames cpu_frames;
        wave_ffmpeg<Scene>(0.0f, n_steps, cpu_frames);
        if (generate_animation<Scene>(cpu_frames, "out/wave_cpu") != 0) {
            std::cout << "CPU animation error: Failed to generate animation."
                      << std::endl;
        } else {
            std::cout << "CPU video has been generated." << std::endl;
        }

        // GPU naive.
        FFmpegFrames gpu_naive_frames;
        wave_ffmpeg_gpu<Scene>(0.0f, n_steps, gpu_naive_frames);
        if (generate_animation<Scene>(gpu_naive_frames, "out/wave_gpu_naive") != 0) {
            std::cout << "GPU_naive animation error: Failed to generate animation."
                      << std::endl;
        } else {
            std::cout << "GPU_naive video has been generated." << std::endl;
        }

        // GPU shared memory.
        FFmpegFrames gpu_shmem_frames;
        wave_ffmpeg_gpu_shmem<Scene>(0.0f, n_steps, gpu_shmem_frames);
        if (generate_animation<Scene>(gpu_shmem_frames, "out/wave_gpu_shmem") != 0) {
            std::cout << "GPU_shem animation error: Failed to generate animation."
                      << std::endl;
        } else {
            std::cout << "GPU_shmem video has been generated." << std::endl;
        }
    }

    return 0;
}
