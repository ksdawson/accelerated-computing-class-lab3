#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <limits>
#include <utility>
#include <vector>

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Already Written)

// 'wave_cpu_step':
//
// Input:
//
//     t -- time coordinate
//     u(t - dt) in array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t) in array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     u(t + dt) in array 'u0' (overwrites the input)
//
template <typename Scene> void wave_cpu_step(float t, float *u0, float const *u1) {
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;
    constexpr float c = Scene::c;
    constexpr float dx = Scene::dx;
    constexpr float dt = Scene::dt;

    for (int32_t idx_y = 0; idx_y < n_cells_y; ++idx_y) {
        for (int32_t idx_x = 0; idx_x < n_cells_x; ++idx_x) {
            int32_t idx = idx_y * n_cells_x + idx_x;
            bool is_border =
                (idx_x == 0 || idx_x == n_cells_x - 1 || idx_y == 0 ||
                 idx_y == n_cells_y - 1);
            float u_next_val;
            if (is_border || Scene::is_wall(idx_x, idx_y)) {
                u_next_val = 0.0f;
            } else if (Scene::is_source(idx_x, idx_y)) {
                u_next_val = Scene::source_value(idx_x, idx_y, t);
            } else {
                constexpr float coeff = c * c * dt * dt / (dx * dx);
                float damping = Scene::damping(idx_x, idx_y);
                u_next_val =
                    ((2.0f - damping - 4.0f * coeff) * u1[idx] -
                     (1.0f - damping) * u0[idx] +
                     coeff *
                         (u1[idx - 1] + u1[idx + 1] + u1[idx - n_cells_x] +
                          u1[idx + n_cells_x]));
            }
            u0[idx] = u_next_val;
        }
    }
}

// 'wave_cpu':
//
// Input:
//
//     t0 -- initial time coordinate
//     n_steps -- number of time steps to simulate
//     u(t0 - dt) in array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t0) in array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     Overwrites contents of memory pointed to by 'u0' and 'u1'
//
//     Returns pointers to buffers containing the final states of the wave
//     u(t0 + (n_steps - 1) * dt) and u(t0 + n_steps * dt).
//
template <typename Scene>
std::pair<float *, float *> wave_cpu(float t0, int32_t n_steps, float *u0, float *u1) {
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step++) {
        float t = t0 + idx_step * Scene::dt;
        wave_cpu_step<Scene>(t, u0, u1);
        std::swap(u0, u1);
    }
    return {u0, u1};
}

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// GPU Implementation (Naive)

// 'wave_gpu_step':
//
// Input:
//
//     t -- time coordinate
//     u(t - dt) in GPU array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t) in GPU array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     u(t + dt) in GPU array 'u0' (overwrites the input)
//
template <typename Scene>
__global__ void wave_gpu_naive_step(
    float t,
    float *u0,      /* pointer to GPU memory */
    float const *u1, /* pointer to GPU memory */
    uint8_t ilp_size = 1
) {
    // Scene parameters
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;
    constexpr float c = Scene::c;
    constexpr float dx = Scene::dx;
    constexpr float dt = Scene::dt;

    // Thread info
    int tot_threads = gridDim.x * blockDim.x;
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Flatten the 2D iteration space into 1D and stride tot_threads pixels each iteration
    for (uint64_t idx = thread_index * ilp_size; idx < n_cells_y * n_cells_x; idx += tot_threads * ilp_size) {
        #pragma unroll
        for (uint8_t i = 0; i < ilp_size; ++i) {
            // Use 32x1 vectors
            uint64_t ilp_idx = idx + i;
            uint32_t idx_y = ilp_idx / n_cells_x;
            uint32_t idx_x = ilp_idx % n_cells_x;

            // Wave math
            bool is_border =
                (idx_x == 0 || idx_x == n_cells_x - 1 || idx_y == 0 ||
                    idx_y == n_cells_y - 1);
            float u_next_val;
            if (is_border || Scene::is_wall(idx_x, idx_y)) {
                u_next_val = 0.0f;
            } else if (Scene::is_source(idx_x, idx_y)) {
                u_next_val = Scene::source_value(idx_x, idx_y, t);
            } else {
                constexpr float coeff = c * c * dt * dt / (dx * dx);
                float damping = Scene::damping(idx_x, idx_y);
                u_next_val =
                    ((2.0f - damping - 4.0f * coeff) * u1[ilp_idx] -
                        (1.0f - damping) * u0[ilp_idx] +
                        coeff *
                            (u1[ilp_idx - 1] + u1[ilp_idx + 1] + u1[ilp_idx - n_cells_x] +
                            u1[ilp_idx + n_cells_x]));
            }
            u0[ilp_idx] = u_next_val;
        }
    }
}

// 'wave_gpu_naive':
//
// Input:
//
//     t0 -- initial time coordinate
//     n_steps -- number of time steps to simulate
//     u(t0 - dt) in GPU array 'u0' of size 'n_cells_y * n_cells_x'
//     u(t0) in GPU array 'u1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     Launches kernels to overwrite the GPU memory pointed to by 'u0' and 'u1'
//
//     Returns pointers to GPU buffers which will contain the final states of
//     the wave u(t0 + (n_steps - 1) * dt) and u(t0 + n_steps * dt) after all
//     launched kernels have completed.
//
template <typename Scene>
std::pair<float *, float *> wave_gpu_naive(
    float t0,
    int32_t n_steps,
    float *u0, /* pointer to GPU memory */
    float *u1  /* pointer to GPU memory */
) {
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step++) {
        float t = t0 + idx_step * Scene::dt;
        wave_gpu_naive_step<Scene><<<48, 32 * 32>>>(t, u0, u1);
        std::swap(u0, u1);
    }
    return {u0, u1};
}

////////////////////////////////////////////////////////////////////////////////
// GPU Implementation (With Shared Memory)

template <typename Scene>
__global__ void wave_gpu_shmem_multistep(
    float t0, uint32_t ti_step, uint32_t tf_step, // Time params
    float *u0, float *u1 // Buffer params
) {
    // Setup the block SRAM
    // extern __shared__ float sram[];

    // Scene parameters
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;
    constexpr float c = Scene::c;
    constexpr float dx = Scene::dx;
    constexpr float dt = Scene::dt;

    // Tile dimensions
    uint8_t tiles_per_col = 8; // Tuning parameter: We want as square as possible tiles?
    uint8_t tiles_per_row = gridDim.x / tiles_per_col;

    // Tile coordinates
    uint8_t tile_j = blockIdx.x / tiles_per_col;
    uint8_t tile_i = blockIdx.x % tiles_per_col;

    // Divide the grid into tiles (valid data that must be written back at the end)
    uint32_t tile_height = n_cells_x / tiles_per_col;
    uint32_t tile_width = n_cells_y / tiles_per_row;

    // Calculate starting global idx of the tile
    uint32_t global_idx_y = tile_j * tile_width;
    uint32_t global_idx_x = tile_i * tile_height;

    // Handle grids not divisible by the number of SMs
    uint8_t extra_rows = n_cells_x % tiles_per_col;
    uint8_t extra_cols = n_cells_y % tiles_per_row;
    // Assign the extra to the edges since they have smaller overlap (limit to last for simplicity)
    tile_width += (tile_j == tiles_per_row - 1) ? extra_cols : 0;
    tile_height += (tile_i == tiles_per_col - 1) ? extra_rows : 0;

    // Expand the tile by the number of time steps in each direction (overlap for invalid data)
    uint8_t time_steps = tf_step - ti_step;
    uint8_t tile_expansion = time_steps - 1; // 1 time step shouldn't expand
    // Edges can only expand in one dir
    tile_width += (tile_j == 0 || tile_j == tiles_per_row - 1) ? tile_expansion : 2 * tile_expansion;
    tile_height += (tile_i == 0 || tile_i == tiles_per_col - 1) ? tile_expansion : 2 * tile_expansion;

    // Debugging
    // if (threadIdx.x == 0) {
        // printf("n_cells_y: %u, n_cells_x: %u\n", n_cells_y, n_cells_x);
        // printf("tile_index: %d, tile_j: %u, tile_i: %u, tile_height: %u, tile_width: %u, tile_size: %u, tpc: %u, tpr: %u\n", blockIdx.x, tile_j, tile_i, tile_height, tile_width, tile_height*tile_width, tiles_per_col, tiles_per_row);
        // printf("tile_index: %d, tile_j: %llu, tile_i: %llu\n", blockIdx.x, tile_j, tile_i);
        // printf("starting_global_idx: %u, global_idx_y: %u, global_idx_x: %u\n", starting_global_idx, global_idx_y, global_idx_x);
        // printf("left: %d, right: %d, top: %d, bottom: %d\n", left, right, top, bottom);
    // }
    // return;

    // Iterate over the time steps
    for (uint32_t idx_step = ti_step; idx_step < tf_step; ++idx_step) {
        // Calculate t
        float t = t0 + idx_step * dt;

        // Flatten the 2D iteration space into 1D and stride tot_threads pixels each iteration
        for (uint64_t local_idx = threadIdx.x; local_idx < tile_height * tile_width; local_idx += blockDim.x) {
            // Re-map local idx to global idx
            uint64_t local_idx_y = local_idx / tile_height;
            uint64_t local_idx_x = local_idx % tile_height;
            uint32_t idx_y = global_idx_y + local_idx_y;
            uint32_t idx_x = global_idx_x + local_idx_x;

            // Wave math
            int32_t idx = idx_y * n_cells_x + idx_x;
            bool is_border =
                (idx_x == 0 || idx_x == n_cells_x - 1 || idx_y == 0 ||
                    idx_y == n_cells_y - 1);
            float u_next_val;
            if (is_border || Scene::is_wall(idx_x, idx_y)) {
                u_next_val = 0.0f;
            } else if (Scene::is_source(idx_x, idx_y)) {
                u_next_val = Scene::source_value(idx_x, idx_y, t);
            } else {
                constexpr float coeff = c * c * dt * dt / (dx * dx);
                float damping = Scene::damping(idx_x, idx_y);
                u_next_val =
                    ((2.0f - damping - 4.0f * coeff) * u1[idx] -
                        (1.0f - damping) * u0[idx] +
                        coeff *
                            (u1[idx - 1] + u1[idx + 1] + u1[idx - n_cells_x] +
                            u1[idx + n_cells_x]));
            }
            u0[idx] = u_next_val;
        }

        // We need the new pixel for all pixels in the block before processing the next time step
        __syncthreads();

        // u0 contains the most recent timestamp and u1 contains the second most recent so swap
        std::swap(u0, u1); // Only swaps pointers in local registers

        // Shrink the tile
        if (tile_j == 0 || tile_j == tiles_per_row - 1) {
            tile_width -=1;
        } else {
            tile_width -= 2;
            ++global_idx_y; // Go over a col
        }
        if (tile_i == 0 || tile_i == tiles_per_col - 1) {
            tile_height -= 1;
        } else {
            tile_height -= 2;
            ++global_idx_x; // Go down a row
        }
    }
}

// 'wave_gpu_shmem':
//
// Input:
//
//     t0 -- initial time coordinate
//
//     n_steps -- number of time steps to simulate
//
//     u(t0 - dt) in GPU array 'u0' of size 'n_cells_y * n_cells_x'
///
//     u(t0) in GPU array 'u1' of size 'n_cells_y * n_cells_x'
//
//     Scratch buffers 'extra0' and 'extra1' of size 'n_cells_y * n_cells_x'
//
// Output:
//
//     Launches kernels to (potentially) overwrite the GPU memory pointed to
//     by 'u0' and 'u1', 'extra0', and 'extra1'.
//
//     Returns pointers to GPU buffers which will contain the final states of
//     the wave u(t0 + (n_steps - 1) * dt) and u(t0 + n_steps * dt) after all
//     launched kernels have completed. These buffers can be any of 'u0', 'u1',
//     'extra0', or 'extra1'.
//
template <typename Scene>
std::pair<float *, float *> wave_gpu_shmem(
    float t0,
    int32_t n_steps,
    float *u0,     /* pointer to GPU memory */
    float *u1,     /* pointer to GPU memory */
    float *extra0, /* pointer to GPU memory */
    float *extra1  /* pointer to GPU memory */
) {
    // Number of time steps to process at once in a kernel
    uint8_t time_steps = 1;

    for (uint32_t idx_step = 0; idx_step < n_steps; idx_step += time_steps) {
        // Compute starting and ending time step
        uint32_t ti_step = idx_step;
        uint32_t tf_step = ti_step + min(n_steps - idx_step, time_steps);

        // Setup the block SRAM
        // int shmem_size_bytes = 100 * 1000; // Max 100 KB per block
        // CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
        //     wave_gpu_shmem_multistep<Scene>),
        //     hipFuncAttributeMaxDynamicSharedMemorySize,
        //     shmem_size_bytes
        // ));

        // Launch our kernel
        // wave_gpu_shmem_multistep<Scene><<<48, 32 * 32, shmem_size_bytes>>>(t0, ti_step, tf_step, u0, u1);
        wave_gpu_shmem_multistep<Scene><<<48, 32 * 32>>>(t0, ti_step, tf_step, u0, u1);

        // Debugging
        // return {u0, u1};

        // On an odd number of time steps we need to swap our pointers
        if (time_steps % 2 != 0) {
            std::swap(u0, u1);
        }
    }
    return {u0, u1};
}

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

struct BaseScene {
    constexpr static int32_t n_cells_x = 3201;
    constexpr static int32_t n_cells_y = 3201;
    constexpr static float c = 1.0f;
    constexpr static float dx = 1.0f / float(n_cells_x - 1);
    constexpr static float dy = 1.0f / float(n_cells_y - 1);
    constexpr static float dt = 0.25f * dx / c;
    constexpr static float t_end = 1.0f;
};

struct BaseSceneSmallScale {
    constexpr static int32_t n_cells_x = 201;
    constexpr static int32_t n_cells_y = 201;
    constexpr static float c = 1.0f;
    constexpr static float dx = 1.0f / float(n_cells_x - 1);
    constexpr static float dy = 1.0f / float(n_cells_y - 1);
    constexpr static float dt = 0.25f * dx / c;
    constexpr static float t_end = 1.0f;
};

float __host__ __device__ __forceinline__ boundary_damping(
    int32_t n_cells_x,
    int32_t n_cells_y,
    float ramp_size,
    float max_damping,
    int32_t idx_x,
    int32_t idx_y) {
    float x = float(idx_x) / (n_cells_x - 1);
    float y = float(idx_y) / (n_cells_y - 1);
    float fx = 1.0f - min(min(x, 1.0f - x), ramp_size) / ramp_size;
    float fy = 1.0f - min(min(y, 1.0f - y), ramp_size) / ramp_size;
    float f = max(fx, fy);
    return max_damping * f * f;
}

struct PointSource : public BaseScene {
    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        return false;
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 2 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 20.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

struct Slit : public BaseScene {
    constexpr static float slit_width = 0.05f;

    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        float y = float(idx_y) / (n_cells_y - 1);
        return idx_x == (n_cells_x - 1) / 2 &&
            (y < 0.5f - slit_width / 2 || y > 0.5f + slit_width / 2);
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 4 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 40.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

struct DoubleSlit : public BaseScene {
    constexpr static float slit_width = 0.03f;

    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        float y = float(idx_y) / (n_cells_y - 1);
        return (idx_x == (n_cells_x - 1) * 2 / 3) &&
            !((y >= 0.45f - slit_width / 2 && y <= 0.45f + slit_width / 2) ||
              (y >= 0.55f - slit_width / 2 && y <= 0.55f + slit_width / 2));
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 6 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 20.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

struct DoubleSlitSmallScale : public BaseSceneSmallScale {
    constexpr static float slit_width = 0.03f;

    static __host__ __device__ __forceinline__ bool
    is_wall(int32_t idx_x, int32_t idx_y) {
        constexpr float EPS = 1e-6;
        float y = float(idx_y) / (n_cells_y - 1);
        return (idx_x == (n_cells_x - 1) * 2 / 3) &&
            !((y >= 0.45f - slit_width / 2 - EPS && y <= 0.45f + slit_width / 2 + EPS) ||
              (y >= 0.55f - slit_width / 2 - EPS && y <= 0.55f + slit_width / 2 + EPS));
    }

    static __host__ __device__ __forceinline__ bool
    is_source(int32_t idx_x, int32_t idx_y) {
        return idx_x == (n_cells_x - 1) / 6 && idx_y == (n_cells_y - 1) / 2;
    }

    static __host__ __device__ __forceinline__ float
    source_value(int32_t idx_x, int32_t idx_y, float t) {
        return 10.0f * sinf(2.0f * 3.14159265359f * 20.0f * t);
    }

    static __host__ __device__ __forceinline__ float
    damping(int32_t idx_x, int32_t idx_y) {
        return boundary_damping(n_cells_x, n_cells_y, 0.1f, 0.5f, idx_x, idx_y);
    }
};

// Output image writers: BMP file header structure
#pragma pack(push, 1)
struct BMPHeader {
    uint16_t fileType{0x4D42};   // File type, always "BM"
    uint32_t fileSize{0};        // Size of the file in bytes
    uint16_t reserved1{0};       // Always 0
    uint16_t reserved2{0};       // Always 0
    uint32_t dataOffset{54};     // Start position of pixel data
    uint32_t headerSize{40};     // Size of this header (40 bytes)
    int32_t width{0};            // Image width in pixels
    int32_t height{0};           // Image height in pixels
    uint16_t planes{1};          // Number of color planes
    uint16_t bitsPerPixel{24};   // Bits per pixel (24 for RGB)
    uint32_t compression{0};     // Compression method (0 for uncompressed)
    uint32_t imageSize{0};       // Size of raw bitmap data
    int32_t xPixelsPerMeter{0};  // Horizontal resolution
    int32_t yPixelsPerMeter{0};  // Vertical resolution
    uint32_t colorsUsed{0};      // Number of colors in the color palette
    uint32_t importantColors{0}; // Number of important colors
};
#pragma pack(pop)

void writeBMP(
    const char *fname,
    uint32_t width,
    uint32_t height,
    const std::vector<uint8_t> &pixels) {
    BMPHeader header;
    header.width = width;
    header.height = height;

    uint32_t rowSize = (width * 3 + 3) & (~3); // Align to 4 bytes
    header.imageSize = rowSize * height;
    header.fileSize = header.dataOffset + header.imageSize;

    std::ofstream file(fname, std::ios::binary);
    file.write(reinterpret_cast<const char *>(&header), sizeof(header));

    // Write pixel data with padding
    std::vector<uint8_t> padding(rowSize - width * 3, 0);
    for (int32_t idx_y = height - 1; idx_y >= 0;
         --idx_y) { // BMP stores pixels from bottom to top
        const uint8_t *row = &pixels[idx_y * width * 3];
        file.write(reinterpret_cast<const char *>(row), width * 3);
        if (!padding.empty()) {
            file.write(reinterpret_cast<const char *>(padding.data()), padding.size());
        }
    }
}

// If trunc - cut the border of the image.
template <typename Scene>
std::vector<uint8_t> render_wave(const float *u, int trunc = 0) {
    constexpr int32_t n_cells_x = Scene::n_cells_x;
    constexpr int32_t n_cells_y = Scene::n_cells_y;

    std::vector<uint8_t> pixels((n_cells_x - trunc) * (n_cells_y - trunc) * 3);
    for (int32_t idx_y = 0; idx_y < n_cells_y - trunc; ++idx_y) {
        for (int32_t idx_x = 0; idx_x < n_cells_x - trunc; ++idx_x) {
            int32_t idx = idx_y * (n_cells_x - trunc) + idx_x;
            int32_t u_idx = idx_y * n_cells_x + idx_x;
            float val = u[u_idx];
            bool is_wall = Scene::is_wall(idx_x, idx_y);
            // BMP stores pixels in BGR order
            if (is_wall) {
                pixels[idx * 3 + 2] = 0;
                pixels[idx * 3 + 1] = 0;
                pixels[idx * 3 + 0] = 0;
            } else if (val > 0.0f) {
                pixels[idx * 3 + 2] = 255;
                pixels[idx * 3 + 1] = 255 - uint8_t(min(val * 255.0f, 255.0f));
                pixels[idx * 3 + 0] = 255 - uint8_t(min(val * 255.0f, 255.0f));
            } else {
                pixels[idx * 3 + 2] = 255 - uint8_t(min(-val * 255.0f, 255.0f));
                pixels[idx * 3 + 1] = 255 - uint8_t(min(-val * 255.0f, 255.0f));
                pixels[idx * 3 + 0] = 255;
            }
        }
    }
    return pixels;
}

struct Results {
    std::vector<float> u0_final;
    std::vector<float> u1_final;
    double time_ms;
};

template <typename Scene, typename F>
Results run_cpu(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    F func) {
    auto u0 = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u1 = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);

    std::pair<float *, float *> u_final;

    double best_time = std::numeric_limits<double>::infinity();
    for (int32_t i = 0; i < num_iters_outer; ++i) {
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t j = 0; j < num_iters_inner; ++j) {
            std::fill(u0.begin(), u0.end(), 0.0f);
            std::fill(u1.begin(), u1.end(), 0.0f);
            u_final = func(t0, n_steps, u0.data(), u1.data());
        }
        auto end = std::chrono::high_resolution_clock::now();
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count() /
            num_iters_inner;
        best_time = std::min(best_time, time_ms);
    }

    if (u_final.first == u1.data() && u_final.second == u0.data()) {
        std::swap(u0, u1);
    } else if (!(u_final.first == u0.data() && u_final.second == u1.data())) {
        std::cerr << "Unexpected return values from 'func'" << std::endl;
        std::abort();
    }

    return {std::move(u0), std::move(u1), best_time};
}

template <typename Scene, typename F>
Results run_gpu(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    bool use_extra,
    F func) {
    float *u0;
    float *u1;
    float *extra0 = nullptr;
    float *extra1 = nullptr;

    CUDA_CHECK(hipMalloc(&u0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&u1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));

    if (use_extra) {
        CUDA_CHECK(
            hipMalloc(&extra0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
        CUDA_CHECK(
            hipMalloc(&extra1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    }

    std::pair<float *, float *> u_final;

    double best_time = std::numeric_limits<double>::infinity();
    for (int32_t i = 0; i < num_iters_outer; ++i) {
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t j = 0; j < num_iters_inner; ++j) {
            CUDA_CHECK(
                hipMemset(u0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
            CUDA_CHECK(
                hipMemset(u1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
            if (use_extra) {
                CUDA_CHECK(hipMemset(
                    extra0,
                    0,
                    Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
                CUDA_CHECK(hipMemset(
                    extra1,
                    0,
                    Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
            }
            u_final = func(t0, n_steps, u0, u1, extra0, extra1);
        }
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count() /
            num_iters_inner;
        best_time = std::min(best_time, time_ms);
    }

    if (u_final.first != u0 && u_final.first != u1 &&
        (extra0 == nullptr || u_final.first != extra0) &&
        (extra1 == nullptr || u_final.first != extra1)) {
        std::cerr << "Unexpected final 'u0' pointer returned from GPU implementation"
                  << std::endl;
        std::abort();
    }

    if (u_final.second != u0 && u_final.second != u1 &&
        (extra0 == nullptr || u_final.second != extra0) &&
        (extra1 == nullptr || u_final.second != extra1)) {
        std::cerr << "Unexpected final 'u1' pointer returned from GPU implementation"
                  << std::endl;
        std::abort();
    }

    auto u0_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u1_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    CUDA_CHECK(hipMemcpy(
        u0_cpu.data(),
        u_final.first,
        Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
        hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(
        u1_cpu.data(),
        u_final.second,
        Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
        hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(u0));
    CUDA_CHECK(hipFree(u1));
    if (use_extra) {
        CUDA_CHECK(hipFree(extra0));
        CUDA_CHECK(hipFree(extra1));
    }

    return {std::move(u0_cpu), std::move(u1_cpu), best_time};
}

template <typename Scene, typename F>
Results run_gpu_extra(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    F func) {
    return run_gpu<Scene>(t0, n_steps, num_iters_outer, num_iters_inner, true, func);
}

template <typename Scene, typename F>
Results run_gpu_no_extra(
    float t0,
    int32_t n_steps,
    int32_t num_iters_outer,
    int32_t num_iters_inner,
    F func) {
    return run_gpu<Scene>(
        t0,
        n_steps,
        num_iters_outer,
        num_iters_inner,
        false,
        [&](float t0,
            int32_t n_steps,
            float *u0,
            float *u1,
            float *extra0,
            float *extra1) { return func(t0, n_steps, u0, u1); });
}

double rel_rmse(std::vector<float> const &a, std::vector<float> const &b) {
    if (a.size() != b.size()) {
        std::cerr << "Mismatched sizes in 'rel_rmse'" << std::endl;
        std::abort();
    }
    double ref_sum = 0.0;
    double sum = 0.0;
    for (size_t i = 0; i < a.size(); ++i) {
        ref_sum += double(a.at(i)) * double(a.at(i));
        double diff = double(a.at(i)) - double(b.at(i));
        sum += diff * diff;
    }
    return sqrt(sum / a.size()) / sqrt(ref_sum / a.size());
}

// FFmpeg implementations.
typedef std::vector<std::vector<uint8_t>> FFmpegFrames;

// CPU implementation with FFmpeg framing.
template <typename Scene>
void wave_ffmpeg(float t0, int32_t n_steps, FFmpegFrames &frames) {
    static constexpr int32_t frame_step = 2;
    auto u0_v = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u1_v = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    auto u0 = u0_v.data();
    auto u1 = u1_v.data();
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step += frame_step) {
        auto r = wave_cpu<Scene>(t0 + idx_step * Scene::dt, frame_step, u0, u1);
        u0 = r.first;
        u1 = r.second;
        frames.push_back(render_wave<Scene>(u1, 1));
    }
}

template <typename Scene>
void wave_ffmpeg_gpu(float t0, int32_t n_steps, FFmpegFrames &frames) {
    static constexpr int32_t frame_step = 2;
    auto u1_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    float *u0;
    float *u1;
    CUDA_CHECK(hipMalloc(&u0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&u1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step += frame_step) {
        auto r = wave_gpu_naive<Scene>(t0 + idx_step * Scene::dt, frame_step, u0, u1);
        u0 = r.first;
        u1 = r.second;
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(
            u1_cpu.data(),
            u1,
            Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
            hipMemcpyDeviceToHost));
        frames.push_back(render_wave<Scene>(u1_cpu.data(), 1));
    }
}

template <typename Scene>
void wave_ffmpeg_gpu_shmem(float t0, int32_t n_steps, FFmpegFrames &frames) {
    static constexpr int32_t frame_step = 2;
    auto u1_cpu = std::vector<float>(Scene::n_cells_x * Scene::n_cells_y);
    float *u0;
    float *u1;
    float *extra0;
    float *extra1;
    CUDA_CHECK(hipMalloc(&u0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&u1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMemset(u1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&extra0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    CUDA_CHECK(hipMalloc(&extra1, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
    float *buffers[] = {u0, u1, extra0, extra1};
    for (int32_t idx_step = 0; idx_step < n_steps; idx_step += frame_step) {
        CUDA_CHECK(
            hipMemset(extra0, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
        CUDA_CHECK(
            hipMemset(extra1, 0, Scene::n_cells_x * Scene::n_cells_y * sizeof(float)));
        auto r = wave_gpu_shmem<Scene>(
            t0 + idx_step * Scene::dt,
            frame_step,
            u0,
            u1,
            extra0,
            extra1);
        u0 = r.first;
        u1 = r.second;
        for (int i = 0; i < 4; ++i) {
            if (buffers[i] != u0 && buffers[i] != u1) {
                extra0 = buffers[i];
            }
        }
        for (int i = 0; i < 4; ++i) {
            if (buffers[i] != u0 && buffers[i] != u1 && buffers[i] != extra0) {
                extra1 = buffers[i];
            }
        }
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(
            u1_cpu.data(),
            u1,
            Scene::n_cells_x * Scene::n_cells_y * sizeof(float),
            hipMemcpyDeviceToHost));
        frames.push_back(render_wave<Scene>(u1_cpu.data(), 1));
    }
}

template <typename Scene>
int generate_animation(const FFmpegFrames &frames, std::string fname) {
    std::string ffmpeg_command = "ffmpeg -y "
                                 "-f rawvideo "
                                 "-pixel_format rgb24 "
                                 "-video_size " +
        std::to_string(Scene::n_cells_x - 1) + "x" +
        std::to_string(Scene::n_cells_y - 1) +
        " "
        "-framerate " +
        std::to_string(30) +
        " "
        "-i - "
        "-c:v libx264 "
        "-pix_fmt yuv420p " +
        fname + ".mp4" + " 2> /dev/null";

    FILE *pipe = popen(ffmpeg_command.c_str(), "w");
    if (!pipe) {
        std::cerr << "Failed to open pipe to FFmpeg." << std::endl;
        return 1;
    }

    for (auto &frame : frames) {
        if (fwrite(frame.data(), 1, frame.size(), pipe) != frame.size()) {
            std::cerr << "Failed to write frame to FFmpeg." << std::endl;
            return 1;
        }
    }

    pclose(pipe);
    return 0;
}

int main(int argc, char **argv) {
    // Small scale tests: mainly for correctness.
    double tolerance = 3e-2;
    bool gpu_naive_correct = false;
    bool gpu_shmem_correct = false;
    {
        printf("Small scale tests (on scene 'DoubleSlitSmallScale'):\n");
        using Scene = DoubleSlitSmallScale;

        // CPU.
        int32_t n_steps = Scene::t_end / Scene::dt;
        auto cpu_results = run_cpu<Scene>(0.0f, n_steps, 1, 1, wave_cpu<Scene>);
        writeBMP(
            "out/wave_cpu_small_scale.bmp",
            Scene::n_cells_x,
            Scene::n_cells_y,
            render_wave<Scene>(cpu_results.u0_final.data()));
        printf("  CPU sequential implementation:\n");
        printf("    run time: %.2f ms\n", cpu_results.time_ms);
        printf("\n");

        // GPU: wave_gpu_naive.
        auto gpu_naive_results =
            run_gpu_no_extra<Scene>(0.0f, n_steps, 1, 1, wave_gpu_naive<Scene>);
        writeBMP(
            "out/wave_gpu_naive_small_scale.bmp",
            Scene::n_cells_x,
            Scene::n_cells_y,
            render_wave<Scene>(gpu_naive_results.u0_final.data()));
        double naive_rel_rmse =
            rel_rmse(cpu_results.u0_final, gpu_naive_results.u0_final);
        if (naive_rel_rmse < tolerance) {
            gpu_naive_correct = true;
        }
        printf("  GPU naive implementation:\n");
        printf("    run time: %.2f ms\n", gpu_naive_results.time_ms);
        printf("    correctness: %.2e relative RMSE\n", naive_rel_rmse);
        printf("\n");

        // GPU: wave_gpu_shmem.
        auto gpu_shmem_results =
            run_gpu_extra<Scene>(0.0f, n_steps, 1, 1, wave_gpu_shmem<Scene>);
        writeBMP(
            "out/wave_gpu_shmem_small_scale.bmp",
            Scene::n_cells_x,
            Scene::n_cells_y,
            render_wave<Scene>(gpu_shmem_results.u0_final.data()));
        double shmem_rel_rmse =
            rel_rmse(cpu_results.u0_final, gpu_shmem_results.u0_final);
        if (shmem_rel_rmse < tolerance) {
            gpu_shmem_correct = true;
        }
        printf("  GPU shared memory implementation:\n");
        printf("    run time: %.2f ms\n", gpu_shmem_results.time_ms);
        printf("    correctness: %.2e relative RMSE\n", shmem_rel_rmse);
        printf("\n");

        if (gpu_naive_correct) {
            printf(
                "  CPU -> GPU naive speedup: %.2fx\n",
                cpu_results.time_ms / gpu_naive_results.time_ms);
        }
        if (gpu_shmem_correct) {
            printf(
                "  CPU -> GPU shared memory speedup: %.2fx\n",
                cpu_results.time_ms / gpu_shmem_results.time_ms);
        }
        if (gpu_naive_correct && gpu_shmem_correct) {
            printf(
                "  GPU naive -> GPU shared memory speedup: %.2fx\n",
                gpu_naive_results.time_ms / gpu_shmem_results.time_ms);
        }
        printf("\n");
    }

    // Run performance tests if requested.
    bool run_perf_tests = false;
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "-p") == 0) {
            run_perf_tests = true;
            break;
        }
    }

    // Large scale tests: mainly for performance.
    if (run_perf_tests) {
        printf("Large scale tests (on scene 'DoubleSlit'):\n");
        using Scene = DoubleSlit;

        int32_t n_steps = Scene::t_end / Scene::dt;
        int32_t num_iters_outer_gpu = 1;
        int32_t num_iters_inner_gpu = 1;

        // GPU: wave_gpu_naive.
        Results gpu_naive_results;
        if (gpu_naive_correct) {
            gpu_naive_results = run_gpu_no_extra<Scene>(
                0.0f,
                n_steps,
                num_iters_outer_gpu,
                num_iters_inner_gpu,
                wave_gpu_naive<Scene>);
            printf("  GPU naive implementation:\n");
            printf("    run time: %.2f ms\n", gpu_naive_results.time_ms);
            printf("\n");
            auto pixels_gpu_naive = render_wave<Scene>(gpu_naive_results.u0_final.data());
            writeBMP(
                "out/wave_gpu_naive_large_scale.bmp",
                Scene::n_cells_x,
                Scene::n_cells_y,
                pixels_gpu_naive);
        } else {
            printf("  Skipping GPU naive implementation (incorrect)\n");
        }

        // GPU: wave_gpu_shmem.
        Results gpu_shmem_results;
        double naive_shmem_rel_rmse = 0.0;
        if (gpu_shmem_correct) {
            gpu_shmem_results = run_gpu_extra<Scene>(
                0.0f,
                n_steps,
                num_iters_outer_gpu,
                num_iters_inner_gpu,
                wave_gpu_shmem<Scene>);
            naive_shmem_rel_rmse =
                rel_rmse(gpu_naive_results.u0_final, gpu_shmem_results.u0_final);
            printf("  GPU shared memory implementation:\n");
            printf("    run time: %.2f ms\n", gpu_shmem_results.time_ms);
            printf(
                "    correctness (w.r.t. GPU naive): %.2e relative RMSE\n",
                naive_shmem_rel_rmse);
            printf("\n");
            auto pixels_gpu_shmem = render_wave<Scene>(gpu_shmem_results.u0_final.data());
            writeBMP(
                "out/wave_gpu_shmem_large_scale.bmp",
                Scene::n_cells_x,
                Scene::n_cells_y,
                pixels_gpu_shmem);
        } else {
            printf("  Skipping GPU shared memory implementation (incorrect)\n");
        }

        if (gpu_naive_correct && gpu_shmem_correct && naive_shmem_rel_rmse < tolerance) {
            printf(
                "  GPU naive -> GPU shared memory speedup: %.2fx\n",
                gpu_naive_results.time_ms / gpu_shmem_results.time_ms);

        } else {
            printf("  GPU naive -> GPU shared memory speedup: N/A (incorrect)\n");
        }
        printf("\n");
    }

    // Generate animation if requested.
    bool a_flag = false;
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "-a") == 0) {
            a_flag = true;
            break;
        }
    }

    if (a_flag) {
        using Scene = DoubleSlitSmallScale;
        int32_t n_steps = Scene::t_end / Scene::dt;

        // CPU.
        FFmpegFrames cpu_frames;
        wave_ffmpeg<Scene>(0.0f, n_steps, cpu_frames);
        if (generate_animation<Scene>(cpu_frames, "out/wave_cpu") != 0) {
            std::cout << "CPU animation error: Failed to generate animation."
                      << std::endl;
        } else {
            std::cout << "CPU video has been generated." << std::endl;
        }

        // GPU naive.
        FFmpegFrames gpu_naive_frames;
        wave_ffmpeg_gpu<Scene>(0.0f, n_steps, gpu_naive_frames);
        if (generate_animation<Scene>(gpu_naive_frames, "out/wave_gpu_naive") != 0) {
            std::cout << "GPU_naive animation error: Failed to generate animation."
                      << std::endl;
        } else {
            std::cout << "GPU_naive video has been generated." << std::endl;
        }

        // GPU shared memory.
        FFmpegFrames gpu_shmem_frames;
        wave_ffmpeg_gpu_shmem<Scene>(0.0f, n_steps, gpu_shmem_frames);
        if (generate_animation<Scene>(gpu_shmem_frames, "out/wave_gpu_shmem") != 0) {
            std::cout << "GPU_shem animation error: Failed to generate animation."
                      << std::endl;
        } else {
            std::cout << "GPU_shmem video has been generated." << std::endl;
        }
    }

    return 0;
}
